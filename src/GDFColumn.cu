/*
 * GDFColumn.cu
 *
 *  Created on: Sep 12, 2018
 *      Author: rqc
 */

#include "GDFColumn.cuh"

gdf_column_cpp::gdf_column_cpp()
{
    column.data = nullptr;
    column.valid = nullptr;
    column.size = 0;
    column.dtype = GDF_invalid;
    column.null_count = 0;
    this->allocated_size_data = 0;
    this->allocated_size_valid = 0;

}

gdf_column_cpp::gdf_column_cpp(void* _data, gdf_valid_type* _valid, gdf_dtype _dtype, size_t _size, gdf_size_type _null_count)
{
    column.data = _data;
    column.valid = _valid;
    column.size = _size;
    column.dtype = _dtype;
    column.null_count = _null_count;
    int byte_width;
    get_column_byte_width(this,&byte_width);
    this->allocated_size_data = size * byte_width;
    this->allocated_size_valid = (((((this->size()+ 7 ) / 8) + 63 ) / 64) * 64);

}

gdf_column_cpp::gdf_column_cpp(gdf_dtype type, size_t num_values, void * input_data, size_t width_per_value)
{
    create_gdf_column(type, num_values, input_data, width_per_value);

}

gdf_column_cpp::gdf_column_cpp(const gdf_column_cpp& col)
{
    column.data = col.column.data;
    column.valid = col.column.valid;
    column.size = col.column.size;
    column.dtype = col.column.dtype;
    column.null_count = col.column.null_count;
    this->allocated_size_data = col.allocated_size_data;
    this->allocated_size_valid = col.allocated_size_valid;
    GDFRefCounter::getInstance()->increment(const_cast<gdf_column*>(&col.column));

}

gdf_column_cpp::gdf_column_cpp(gdf_column_cpp& col)
{
    column.data = col.column.data;
    column.valid = col.column.valid;
    column.size = col.column.size;
    column.dtype = col.column.dtype;
    column.null_count = col.column.null_count;
    this->allocated_size_data = col.allocated_size_data;
    this->allocated_size_valid = col.allocated_size_valid;
    GDFRefCounter::getInstance()->increment(const_cast<gdf_column*>(&col.column));

}

void gdf_column_cpp::operator=(const gdf_column_cpp& col)
{
    column.data = col.column.data;
    column.valid = col.column.valid;
    column.size = col.column.size;
    column.dtype = col.column.dtype;
    column.null_count = col.column.null_count;
    this->allocated_size_data = col.allocated_size_data;
    this->allocated_size_valid = col.allocated_size_valid;
    GDFRefCounter::getInstance()->increment(const_cast<gdf_column*>(&col.column));

}

gdf_column* gdf_column_cpp::get_gdf_column()
{
    return &column;
}

//TODO: needs to be implemented for efficiency though not strictly necessary
void gdf_column_cpp::compact(){
    if( this->allocation_size_valid != (((((this->size()+ 7 ) / 8) + 63 ) / 64) * 64)){
    	//compact valid allcoation

    }

    int byte_width;
    get_column_byte_width(this,&byte_width);
    if(this->allocated_size_data != (this->size() * byte_width)){
    	//compact data allocation
    }
}

void gdf_column_cpp::create_gdf_column(gdf_dtype type, size_t num_values, void * input_data, size_t width_per_value)
{
    assert(type != GDF_invalid);

    char * data;
    gdf_valid_type * valid_device;

    this->allocation_size_valid = ((((num_values + 7 ) / 8) + 63 ) / 64) * 64; //so allocations are supposed to be 64byte aligned

    hipMalloc((void **) &valid_device, allocation_size_valid);

    hipMemset(valid_device, (gdf_valid_type)255, allocation_size_valid); //assume all relevant bits are set to on

    this->allocated_size_data = width_per_value * num_values;
    hipMalloc((void **) &data, this->allocated_size_data);

    gdf_column_view(&this->column, (void *) data, valid_device, num_values, type);

    if(input_data != nullptr){
        hipMemcpy(data, input_data, num_values * width_per_value, hipMemcpyHostToDevice);
    }


    GDFRefCounter::getInstance()->register_column(&this->column);

}

void gdf_column_cpp::realloc_gdf_column(gdf_dtype type, size_t size, size_t width){
    GDFRefCounter::getInstance()->decrement(&this->column); //decremeting reference, deallocating space

	create_gdf_column(type, size, nullptr, width);
}

gdf_error gdf_column_cpp::gdf_column_view(gdf_column *column, void *data, gdf_valid_type *valid, gdf_size_type size, gdf_dtype dtype)
{
    column->data = data;
    column->valid = valid;
    column->size = size;
    column->dtype = dtype;
    column->null_count = 0;
    return GDF_SUCCESS;
}

gdf_column_cpp::~gdf_column_cpp()
{
    GDFRefCounter::getInstance()->decrement(&this->column);
}
bool gdf_column_cpp::is_ipc(){
	return GDFRefCounter::getInstance()->contains_column(this->data());
}
void* gdf_column_cpp::data(){
    return column.data;
}

gdf_valid_type* gdf_column_cpp::valid(){
    return column.valid;
}
gdf_size_type gdf_column_cpp::size(){
    return column.size;
}

gdf_dtype gdf_column_cpp::dtype(){
    return column.dtype;
}

gdf_size_type gdf_column_cpp::null_count(){
    return column.null_count;
}

gdf_dtype_extra_info gdf_column_cpp::dtype_info(){
    return column.dtype_info;
}

void gdf_column_cpp::set_dtype(gdf_dtype dtype){
    column.dtype=dtype;
}
