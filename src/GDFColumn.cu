/*
 * GDFColumn.cu
 *
 *  Created on: Sep 12, 2018
 *      Author: rqc
 */

#include "GDFColumn.cuh"

#include <gdf/utils.h>

gdf_column_cpp::gdf_column_cpp()
{
    column.data = nullptr;
    column.valid = nullptr;
    column.size = 0;
    column.dtype = GDF_invalid;
    column.null_count = 0;
    this->allocated_size_data = 0;
    this->allocated_size_valid = 0;

}

gdf_column_cpp::gdf_column_cpp(void* _data, gdf_valid_type* _valid, gdf_dtype _dtype, size_t _size, gdf_size_type _null_count, const std::string &column_name)
{
    column.data = _data;
    column.valid = _valid;
    column.size = _size;
    column.dtype = _dtype;
    column.null_count = _null_count;
    int byte_width;
    get_column_byte_width(&column,&byte_width);
    this->allocated_size_data = _size * byte_width;
    //Todo: To validate valid_size
    this->allocated_size_valid = gdf_get_num_chars_bitmask(_size);
    this->column_name = column_name;

}

gdf_column_cpp::gdf_column_cpp(const gdf_column_cpp& col)
{
    column.data = col.column.data;
    column.valid = col.column.valid;
    column.size = col.column.size;
    column.dtype = col.column.dtype;
    column.null_count = col.column.null_count;
    this->allocated_size_data = col.allocated_size_data;
    this->allocated_size_valid = col.allocated_size_valid;
    this->column_name= col.column_name;
    GDFRefCounter::getInstance()->increment(const_cast<gdf_column*>(&col.column));

}

gdf_column_cpp::gdf_column_cpp(gdf_column_cpp& col)
{
    column.data = col.column.data;
    column.valid = col.column.valid;
    column.size = col.column.size;
    column.dtype = col.column.dtype;
    column.null_count = col.column.null_count;
    this->allocated_size_data = col.allocated_size_data;
    this->allocated_size_valid = col.allocated_size_valid;
    this->column_name= col.column_name;
    GDFRefCounter::getInstance()->increment(const_cast<gdf_column*>(&col.column));

}

gdf_column_cpp gdf_column_cpp::clone()  // TODO clone needs to register
{
	void* data_dev = nullptr;
	void* valid_dev = nullptr;

	CheckCudaErrors(hipMalloc(&data_dev, this->allocated_size_data));
	CheckCudaErrors(hipMemcpy(data_dev, this->column.data, this->allocated_size_data, hipMemcpyDeviceToDevice));

	print_gdf_column(this->get_gdf_column());
    if (this->column.valid != nullptr) {
    	CheckCudaErrors(hipMalloc(&valid_dev, this->allocated_size_valid));
	    CheckCudaErrors(hipMemcpy(valid_dev, this->column.valid, this->allocated_size_valid, hipMemcpyDeviceToDevice));
    }

	gdf_column_cpp col1(data_dev,
						(gdf_valid_type*) valid_dev,
						this->column.dtype,
						this->column.size,
						this->column.null_count,
						this->column_name);

//	print_gdf_column(col1.get_gdf_column());

	GDFRefCounter::getInstance()->register_column(&col1.column);

	return col1;
}

void gdf_column_cpp::operator=(const gdf_column_cpp& col)
{
    column.data = col.column.data;
    column.valid = col.column.valid;
    column.size = col.column.size;
    column.dtype = col.column.dtype;
    column.null_count = col.column.null_count;
    this->allocated_size_data = col.allocated_size_data;
    this->allocated_size_valid = col.allocated_size_valid;
    this->column_name= col.column_name;
    GDFRefCounter::getInstance()->increment(const_cast<gdf_column*>(&col.column));

}

gdf_column* gdf_column_cpp::get_gdf_column()
{
    return &column;
}

//TODO: needs to be implemented for efficiency though not strictly necessary
gdf_error gdf_column_cpp::compact(){
    if( this->allocated_size_valid != (((((this->size()+ 7 ) / 8) + 63 ) / 64) * 64)){
    	//compact valid allcoation

    }

    int byte_width;
    get_column_byte_width(this->get_gdf_column(),&byte_width);
    if(this->allocated_size_data != (this->size() * byte_width)){
    	//compact data allocation
    }
}

void gdf_column_cpp::create_gdf_column(gdf_dtype type, size_t num_values, void * input_data, size_t width_per_value, const std::string &column_name)
{
    assert(type != GDF_invalid);

    this->column_name = column_name;

    char * data;
    gdf_valid_type * valid_device;

    this->allocated_size_valid = ((((num_values + 7 ) / 8) + 63 ) / 64) * 64; //so allocations are supposed to be 64byte aligned

    CheckCudaErrors(hipMalloc(&valid_device, allocated_size_valid));

    CheckCudaErrors(hipMemset(valid_device, (gdf_valid_type)255, allocated_size_valid)); //assume all relevant bits are set to on

    this->allocated_size_data = (((width_per_value * num_values) + 63) /64) * 64;
    CheckCudaErrors(hipMalloc(&data, this->allocated_size_data));

    gdf_column_view(&this->column, (void *) data, valid_device, num_values, type);

    if(input_data != nullptr){
        CheckCudaErrors(hipMemcpy(data, input_data, num_values * width_per_value, hipMemcpyHostToDevice));
    }


    GDFRefCounter::getInstance()->register_column(&this->column);

}

void gdf_column_cpp::realloc_gdf_column(gdf_dtype type, size_t size, size_t width){
	const std::string col_name = this->column_name;
    GDFRefCounter::getInstance()->decrement(&this->column); //decremeting reference, deallocating space

	this->create_gdf_column(type, size, nullptr, width, col_name);
}

gdf_error gdf_column_cpp::gdf_column_view(gdf_column *column, void *data, gdf_valid_type *valid, gdf_size_type size, gdf_dtype dtype)
{
    column->data = data;
    column->valid = valid;
    column->size = size;
    column->dtype = dtype;
    column->null_count = 0;
    return GDF_SUCCESS;
}

gdf_column_cpp::~gdf_column_cpp()
{
    GDFRefCounter::getInstance()->decrement(&this->column);
}
bool gdf_column_cpp::is_ipc(){
	return !GDFRefCounter::getInstance()->contains_column(std::make_pair(this->data(),this->valid()));
}
void* gdf_column_cpp::data(){
    return column.data;
}

gdf_valid_type* gdf_column_cpp::valid(){
    return column.valid;
}
gdf_size_type gdf_column_cpp::size(){
    return column.size;
}

gdf_dtype gdf_column_cpp::dtype(){
    return column.dtype;
}

gdf_size_type gdf_column_cpp::null_count(){
    return column.null_count;
}

gdf_dtype_extra_info gdf_column_cpp::dtype_info(){
    return column.dtype_info;
}

void gdf_column_cpp::set_dtype(gdf_dtype dtype){
    column.dtype=dtype;
}
