/*
 * GDFColumn.cu
 *
 *  Created on: Sep 12, 2018
 *      Author: rqc
 */

#include "GDFColumn.cuh"

gdf_column_cpp::gdf_column_cpp()
{
    column.data = nullptr;
    column.valid = nullptr;
    column.size = 0;
    column.dtype = GDF_invalid;
    column.null_count = 0;
    this->allocated_size_data = 0;
    this->allocated_size_valid = 0;
    this->arrived_via_ipc = false;
}

gdf_column_cpp::gdf_column_cpp(void* _data, gdf_valid_type* _valid, gdf_dtype _dtype, size_t _size, gdf_size_type _null_count)
{
    column.data = _data;
    column.valid = _valid;
    column.size = _size;
    column.dtype = _dtype;
    column.null_count = _null_count;
    int byte_width;
    get_column_byte_width(this,&byte_width);
    this->allocated_size_data = size * byte_width;
    this->allocated_size_valid = (((((this->size()+ 7 ) / 8) + 63 ) / 64) * 64);
    this->arrived_via_ipc = false;
}

gdf_column_cpp::gdf_column_cpp(gdf_dtype type, size_t num_values, void * input_data, size_t width_per_value)
{
    create_gdf_column(type, num_values, input_data, width_per_value);
    this->arrived_via_ipc = false;
}

gdf_column_cpp::gdf_column_cpp(const gdf_column_cpp& col)
{
    column.data = col.column.data;
    column.valid = col.column.valid;
    column.size = col.column.size;
    column.dtype = col.column.dtype;
    column.null_count = col.column.null_count;
    this->allocated_size_data = col.allocated_size_data;
    this->allocated_size_valid = col.allocated_size_valid;
    GDFRefCounter::getInstance()->increment(const_cast<gdf_column*>(&col.column));
    this->arrived_via_ipc = col.arrived_via_ipc;
}

gdf_column_cpp::gdf_column_cpp(gdf_column_cpp& col)
{
    column.data = col.column.data;
    column.valid = col.column.valid;
    column.size = col.column.size;
    column.dtype = col.column.dtype;
    column.null_count = col.column.null_count;
    this->allocated_size_data = col.allocated_size_data;
    this->allocated_size_valid = col.allocated_size_valid;
    GDFRefCounter::getInstance()->increment(const_cast<gdf_column*>(&col.column));
    this->arrived_via_ipc = col.arrived_via_ipc;
}

void gdf_column_cpp::operator=(const gdf_column_cpp& col)
{
    column.data = col.column.data;
    column.valid = col.column.valid;
    column.size = col.column.size;
    column.dtype = col.column.dtype;
    column.null_count = col.column.null_count;
    this->allocated_size_data = col.allocated_size_data;
    this->allocated_size_valid = col.allocated_size_valid;
    GDFRefCounter::getInstance()->increment(const_cast<gdf_column*>(&col.column));
    this->arrived_via_ipc = col.arrived_via_ipc;
}

gdf_column* gdf_column_cpp::get_gdf_column()
{
    return &column;
}

//TODO: needs to be implemented for efficiency though not strictly necessary
void gdf_column_cpp::compact(){
    if( this->allocation_size_valid != (((((this->size()+ 7 ) / 8) + 63 ) / 64) * 64)){
    	//compact valid allcoation

    }

    int byte_width;
    get_column_byte_width(this,&byte_width);
    if(this->allocated_size_data != (this->size() * byte_width)){
    	//compact data allocation
    }
}

void gdf_column_cpp::create_gdf_column(gdf_dtype type, size_t num_values, void * input_data, size_t width_per_value)
{
    assert(type != GDF_invalid);

    char * data;
    gdf_valid_type * valid_device;

    this->allocation_size_valid = ((((num_values + 7 ) / 8) + 63 ) / 64) * 64; //so allocations are supposed to be 64byte aligned

    hipMalloc((void **) &valid_device, allocation_size_valid);

    hipMemset(valid_device, (gdf_valid_type)255, allocation_size_valid); //assume all relevant bits are set to on

    this->allocated_size_data = width_per_value * num_values;
    hipMalloc((void **) &data, this->allocated_size_data);

    gdf_column_view(&this->column, (void *) data, valid_device, num_values, type);

    if(input_data != nullptr){
        hipMemcpy(data, input_data, num_values * width_per_value, hipMemcpyHostToDevice);
    }
    this->arrived_via_ipc = false;

    GDFRefCounter::getInstance()->register_column(&this->column);

}

void gdf_column_cpp::realloc_gdf_column(gdf_dtype type, size_t size, size_t width){
    GDFRefCounter::getInstance()->decrement(&this->column); //decremeting reference, deallocating space

	create_gdf_column(type, size, nullptr, width);
}

gdf_error gdf_column_cpp::gdf_column_view(gdf_column *column, void *data, gdf_valid_type *valid, gdf_size_type size, gdf_dtype dtype)
{
    column->data = data;
    column->valid = valid;
    column->size = size;
    column->dtype = dtype;
    column->null_count = 0;
    return GDF_SUCCESS;
}

gdf_column_cpp::~gdf_column_cpp()
{
    GDFRefCounter::getInstance()->decrement(&this->column);
}

void* gdf_column_cpp::data(){
    return column.data;
}

gdf_valid_type* gdf_column_cpp::valid(){
    return column.valid;
}
gdf_size_type gdf_column_cpp::size(){
    return column.size;
}

gdf_dtype gdf_column_cpp::dtype(){
    return column.dtype;
}

gdf_size_type gdf_column_cpp::null_count(){
    return column.null_count;
}

gdf_dtype_extra_info gdf_column_cpp::dtype_info(){
    return column.dtype_info;
}

void gdf_column_cpp::set_dtype(gdf_dtype dtype){
    column.dtype=dtype;
}
