/*
 * GDFCounter.cu
 *
 *  Created on: Sep 12, 2018
 *      Author: rqc
 */
 
 #include "GDFCounter.cuh"

 void GDFRefCounter::increment(key_type key_ptr)
 {
     std::lock_guard<std::mutex> lock(gc_mutex);
 
     if(map.find(key_ptr)==map.end())
         map[key_ptr]=1;
     else
         map[key_ptr]++;
 }
 
 void GDFRefCounter::decrement(key_type key_ptr)
 {
     std::lock_guard<std::mutex> lock(gc_mutex);
     map[key_ptr]--;
 
     if(map[key_ptr]==0)
     {
         map.erase(key_ptr);
         hipFree(key_ptr->data);
         hipFree(key_ptr->valid);
         free(key_ptr);
     }
 }
