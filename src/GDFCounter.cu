/*
 * GDFCounter.cu
 *
 *  Created on: Sep 12, 2018
 *      Author: rqc
 */
 
 #include "GDFCounter.cuh"
 #include <iostream>

GDFRefCounter* GDFRefCounter::Instance=0;

void GDFRefCounter::register_column(gdf_column* col_ptr){

    if(col_ptr != nullptr){
        std::lock_guard<std::mutex> lock(gc_mutex);
        rc_key_t map_key = {col_ptr->data, col_ptr->valid};
        
        if(map.find(map_key) == map.end()){
            map[map_key]=1;
        }
    }
}

void GDFRefCounter::deregister_column(gdf_column* col_ptr)
{
    std::lock_guard<std::mutex> lock(gc_mutex);
    rc_key_t map_key = {col_ptr->data, col_ptr->valid};

    if(map.find(map_key) != map.end()){
        map[map_key]=0; //deregistering
    }
}

void GDFRefCounter::free_if_deregistered(gdf_column* col_ptr)
{
    std::lock_guard<std::mutex> lock(gc_mutex);
    rc_key_t map_key = {col_ptr->data, col_ptr->valid};

    if(map.find(map_key)!=map.end()){
        if(map[map_key]==0){
            map.erase(map_key);
            hipFree(map_key.first); //data
            hipFree(map_key.second); //valid
        }
    }
}

void GDFRefCounter::increment(gdf_column* col_ptr)
{
    std::lock_guard<std::mutex> lock(gc_mutex);
    rc_key_t map_key = {col_ptr->data, col_ptr->valid};

    if(map.find(map_key)!=map.end()){
        if(map[map_key]!=0){ //is already deregistered
            map[map_key]++;
        }
    }
}

void GDFRefCounter::decrement(gdf_column* col_ptr)
{
    std::lock_guard<std::mutex> lock(gc_mutex);
    rc_key_t map_key = {col_ptr->data, col_ptr->valid};

    if(map.find(map_key)!=map.end()){
        if(map[map_key]>0){
            map[map_key]--;

            if(map[map_key]==0){
                map.erase(map_key);
                hipFree(map_key.first); //data
                hipFree(map_key.second); //valid
            }
        }
    }
}

bool GDFRefCounter::contains_column(void * ptr){
	if(this->map.find(ptr) == this->map.end()){
		return false;
	}
	return true;
}

GDFRefCounter::GDFRefCounter()
{

}

// Testing purposes
size_t GDFRefCounter::get_map_size()
{
    return map.size();
}

GDFRefCounter* GDFRefCounter::getInstance()
{
    if(!Instance)
        Instance=new GDFRefCounter();
    return Instance;
}
