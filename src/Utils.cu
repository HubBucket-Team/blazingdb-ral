#include "Utils.cuh"
#include "cuDF/Allocator.h"

void free_gdf_column(gdf_column * column){
	std::cout<<"Deallocating "<<column->data<<std::flush<<"\n";

    try {
        cuDF::Allocator::deallocate(column->data);
        cuDF::Allocator::deallocate(column->valid);
    }
    catch (const cuDF::Allocator::Exception& exception) {
        std::cerr << exception.what() << std::endl;
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }
	//delete column;
	//column = nullptr;
}

void update_null_count(gdf_column * column)
{
    int count;
    gdf_error result = gdf_count_nonzero_mask(column->valid, column->size, &count);
    assert(result == GDF_SUCCESS);
    column->null_count = column->size - static_cast<gdf_size_type>(count);
}