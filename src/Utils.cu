#include "Utils.cuh"
#include "cuDF/Allocator.h"

void free_gdf_column(gdf_column * column){
	std::cout<<"Deallocating "<<column->data<<std::flush<<"\n";

    try {
        cuDF::Allocator::deallocate(column->data);
        cuDF::Allocator::deallocate(column->valid);
    }
    catch (const cuDF::Allocator::Exception& exception) {
        std::cerr << exception.what() << std::endl;
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }
	//delete column;
	//column = nullptr;
}
