#include "Utils.cuh"

void print_column(gdf_column * column){

	char * host_data_out = new char[column->size];
	char * host_valid_out;

	if(column->size % 8 != 0){
		host_valid_out = new char[(column->size + (8 - (column->size % 8)))/8];
	}else{
		host_valid_out = new char[column->size / 8];
	}

	hipMemcpy(host_data_out,column->data,sizeof(int8_t) * column->size, hipMemcpyDeviceToHost);
	hipMemcpy(host_valid_out,column->valid,sizeof(int8_t) * (column->size + GDF_VALID_BITSIZE - 1) / GDF_VALID_BITSIZE, hipMemcpyDeviceToHost);

	std::cout<<"Printing Column"<<std::endl;

	for(int i = 0; i < column->size; i++){
		int col_position = i / 8;
		int bit_offset = 8 - (i % 8);
		std::cout<<"host_data_out["<<i<<"] = "<<((int)host_data_out[i])<<" valid="<<((host_valid_out[col_position] >> bit_offset ) & 1)<<std::endl;
	}

	delete[] host_data_out;
	delete[] host_valid_out;

	std::cout<<std::endl<<std::endl;
}

void free_gdf_column(gdf_column * column){
	hipFree(column->data);
	hipFree(column->valid);
	//delete column;
	//column = nullptr;
}

void realloc_gdf_column(gdf_column * column, size_t size, size_t width){
	free_gdf_column(column);

	create_gdf_column(column,column->dtype,size,nullptr,width);
}

void create_gdf_column(gdf_column * column, gdf_dtype type, size_t num_values, void * input_data, size_t width_per_value){
	char * data;
	gdf_valid_type * valid_device;

	//so allocations are supposed to be 64byte aligned
	size_t allocation_size_valid = ((((num_values + 7 ) / 8) + 63 ) / 64) * 64;
	hipError_t cuda_error = hipMalloc((void **) &valid_device, allocation_size_valid);

	//assume all relevant bits are set to on
	thrust::constant_iterator<unsigned char> valid(255);
	thrust::device_vector<unsigned char> tester(allocation_size_valid);
	thrust::copy(valid, valid + allocation_size_valid, tester.begin());

	thrust::copy(thrust::cuda::par,valid, valid + allocation_size_valid, thrust::detail::make_normal_iterator(valid_device) );
	cuda_error = hipMalloc((void **) &data,width_per_value * num_values);

	gdf_error error = gdf_column_view(column,(void *) data, valid_device,num_values,type);
	if(input_data != nullptr){
		hipMemcpy(data,input_data, num_values * width_per_value, hipMemcpyHostToDevice);
	}

	column->null_count = 0;
}
