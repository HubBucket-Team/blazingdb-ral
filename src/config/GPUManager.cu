#include <hip/hip_runtime.h>
#include <exception>
#include "GPUManager.cuh"
#include "Utils.cuh"


namespace ral {
namespace config {

GPUManager::GPUManager() : currentDeviceId{0} {
  CheckCudaErrors( hipGetDeviceCount(&totalDevices) );
}

GPUManager& GPUManager::getInstance() {
  static GPUManager instance;
  return instance;
}

void GPUManager::initialize(int deviceId) {
  if (deviceId < 0 || deviceId >= totalDevices) {
    throw std::runtime_error("In GPUManager::initialize function: Invalid deviceId");
  }

  currentDeviceId = deviceId;
}

void GPUManager::setDevice() {
  CheckCudaErrors( hipSetDevice(currentDeviceId) );
}

}  // namespace config
}  // namespace ral
