#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : testing-libgdf.cu
 Author      : felipe
 Version     :
 Copyright   : Your copyright notice
 Description : MVP
 ============================================================================
 */

#include <hip/hip_runtime.h>
#include <memory>
#include <algorithm>
#include <thread>
#include "CalciteInterpreter.h"
#include "ResultSetRepository.h"
#include "DataFrame.h"
#include "Utils.cuh"
#include "Types.h"
#include <hip/hip_runtime.h>

#include "FreeMemory.h"

#include "gdf_wrapper/gdf_wrapper.cuh"

#include <tuple>

#include <blazingdb/protocol/api.h>
#include <blazingdb/protocol/message/messages.h>
#include <blazingdb/protocol/message/interpreter/messages.h>
#include <blazingdb/protocol/message/io/file_system.h>
#include "ral-message.cuh"


using namespace blazingdb::protocol;

#include <blazingdb/io/Util/StringUtil.h>

#include <blazingdb/io/FileSystem/HadoopFileSystem.h>
#include <blazingdb/io/FileSystem/S3FileSystem.h>
#include <blazingdb/io/FileSystem/FileSystemRepository.h>
#include <blazingdb/io/FileSystem/FileSystemCommandParser.h>
#include <blazingdb/io/FileSystem/FileSystemManager.h>
#include <blazingdb/io/Config/BlazingContext.h>
#include <blazingdb/io/Library/Logging/Logger.h>
#include <blazingdb/io/Library/Logging/CoutOutput.h>
#include <blazingdb/io/Library/Logging/FileOutput.h>
#include "blazingdb/io/Library/Logging/ServiceLogging.h"

#include "CalciteExpressionParsing.h"
#include "io/data_parser/CSVParser.h"
#include "io/data_parser/ParquetParser.h"

#include "io/data_provider/UriDataProvider.h"
#include "io/data_parser/DataParser.h"
#include "io/data_provider/DataProvider.h"
#include "io/DataLoader.h"


#include "Config/Config.h"

#include "CodeTimer.h"

const Path FS_NAMESPACES_FILE("/tmp/file_system.bin");
using result_pair = std::pair<Status, std::shared_ptr<flatbuffers::DetachedBuffer>>;
using FunctionType = result_pair (*)(uint64_t, Buffer&& buffer);

//TODO percy c.gonzales fix this later
std::string global_ip;
int global_port;

static result_pair  registerFileSystem(uint64_t accessToken, Buffer&& buffer) {
  std::cout << "registerFileSystem: " << accessToken << std::endl;
  blazingdb::message::io::FileSystemRegisterRequestMessage message(buffer.data());

  FileSystemConnection fileSystemConnection;
  Path root("/");
  const std::string authority =  message.getAuthority();
  if (message.isLocal()) {
    fileSystemConnection = FileSystemConnection(FileSystemType::LOCAL);
  } else if (message.isHdfs()) {
    auto hdfs = message.getHdfs();
    fileSystemConnection = FileSystemConnection(hdfs.host, hdfs.port, hdfs.user, (HadoopFileSystemConnection::DriverType)hdfs.driverType, hdfs.kerberosTicket);
  } else if (message.isS3()) {
    auto s3 = message.getS3();
    fileSystemConnection = FileSystemConnection(s3.bucketName, ( S3FileSystemConnection::EncryptionType )s3.encryptionType, s3.kmsKeyAmazonResourceName, s3.accessKeyId, s3.secretKey, s3.sessionToken);
  }
  root = message.getRoot();
  if (root.isValid() == false) {
    std::cout << "something went wrong when registering filesystem ..." << std::endl;
    ResponseErrorMessage errorMessage{ std::string{ "ERROR: Invalid root provided when registering file system"} };
    return std::make_pair(Status_Error, errorMessage.getBufferData());
  }
  FileSystemEntity fileSystemEntity(authority, fileSystemConnection, root);
  bool ok = BlazingContext::getInstance()->getFileSystemManager()->deregisterFileSystem(authority);
  ok = BlazingContext::getInstance()->getFileSystemManager()->registerFileSystem(fileSystemEntity);
	if (ok) { // then save the fs
		const FileSystemRepository fileSystemRepository(FS_NAMESPACES_FILE, true);
		const bool saved = fileSystemRepository.add(fileSystemEntity);
		if (saved == false) {
			std::cout << "WARNING: could not save the registered file system into ... the data file uri ..."; //TODO percy error message
		}
	} else {
   	  std::cout << "something went wrong when registering filesystem ..." << std::endl;
      ResponseErrorMessage errorMessage{ std::string{"ERROR: Something went wrong when registering file system"} };
      return std::make_pair(Status_Error, errorMessage.getBufferData());
	}
  ZeroMessage response{};
  return std::make_pair(Status_Success, response.getBufferData());
}

static result_pair  deregisterFileSystem(uint64_t accessToken, Buffer&& buffer)  {
  std::cout << "deregisterFileSystem: " << accessToken << std::endl;
  blazingdb::message::io::FileSystemDeregisterRequestMessage message(buffer.data());
  auto authority =  message.getAuthority();
  if (authority.empty() == true) {
     ResponseErrorMessage errorMessage{ std::string{"derigistering an empty authority"} };
     return std::make_pair(Status_Error, errorMessage.getBufferData());
  }
  const bool ok = BlazingContext::getInstance()->getFileSystemManager()->deregisterFileSystem(authority);
  if (ok) { // then save the fs
    const FileSystemRepository fileSystemRepository(FS_NAMESPACES_FILE, true);
    const bool deleted = fileSystemRepository.deleteByAuthority(authority);
    if (deleted == false) {
      std::cout << "WARNING: could not delete the registered file system into ... the data file uri ..."; //TODO percy error message
    }
  }
  ZeroMessage response{};
  return std::make_pair(Status_Success, response.getBufferData());
}


query_token_t loadParquetAndInsertToResultRepository(std::string path, connection_id_t connection, bool schema_only) {
	std::cout<<"loadParquet\n";

	query_token_t token = result_set_repository::get_instance().register_query(connection); //register the query so we can receive result requests for it
  Path  csvFile(path);
	Uri directory(csvFile.getParentPath().toString());
  const bool existsParquetDir = BlazingContext::getInstance()->getFileSystemManager()->exists(directory);
	const bool isS3Dir = (directory.getFileSystemType() == FileSystemType::S3);
	if ((existsParquetDir == false) && (isS3Dir == false)) {
    auto error = "parquet table folder doesn't exists: " + directory.toString(false);
    throw std::runtime_error{error};
  }

	std::thread t = std::thread([=]{
		std::vector<Uri> uris(1);
		uris[0] = Uri(path);

		auto provider = ral::io::uri_data_provider(uris);
		auto parser = ral::io::parquet_parser();
	  ral::io::data_loader loader(&parser, &provider);

	  try
	  {
	    CodeTimer blazing_timer;
	    std::vector<gdf_column_cpp> columns;
	    loader.load_data(columns, {}, false);

      blazing_frame output_frame;
      output_frame.add_table(columns);

      double duration = blazing_timer.getDuration();

      result_set_repository::get_instance().update_token(token, output_frame, duration);
    } catch(const std::exception& e) {
      std::cerr << e.what() << '\n';
      result_set_repository::get_instance().update_token(token, blazing_frame{}, 0.0, e.what());
    }
	});
	t.detach();
	return token;
}

static result_pair loadParquetSchema(uint64_t accessToken, Buffer&& buffer) {

  blazingdb::message::io::LoadParquetFileRequestMessage message(buffer.data());

  uint64_t resultToken = 0L;
  try {
    // @todo, what about other parameters
    bool schema_only = true;
    resultToken = loadParquetAndInsertToResultRepository(message.fileSchema()->path, accessToken, schema_only);

  } catch (const std::exception& e) {
     std::cerr << e.what() << std::endl;
     ResponseErrorMessage errorMessage{ std::string{e.what()} };
     return std::make_pair(Status_Error, errorMessage.getBufferData());
  }
  interpreter::NodeConnectionDTO nodeInfo {
      .port = global_port,
      .path = "/tmp/ral.socket",
      .type = NodeConnectionType {NodeConnectionType_TCP}
  };
  interpreter::ExecutePlanResponseMessage responsePayload{resultToken, nodeInfo};
  return std::make_pair(Status_Success, responsePayload.getBufferData());
}

query_token_t loadCsvAndInsertToResultRepository(std::string path, std::vector<std::string> names, std::vector<gdf_dtype> dtypes, std::string delimiter, std::string line_terminator, 
  int skip_rows, connection_id_t connection, bool schema_only) {
	std::cout<<"loadCsv\n";

	query_token_t token = result_set_repository::get_instance().register_query(connection); //register the query so we can receive result requests for it
  Path  csvFile(path);
	Uri directory(csvFile.getParentPath().toString());
  const bool existsParquetDir = BlazingContext::getInstance()->getFileSystemManager()->exists(directory);
	const bool isS3Dir = (directory.getFileSystemType() == FileSystemType::S3);
	if ((existsParquetDir == false) && (isS3Dir == false)) {
    auto error = "csv table folder doesn't exists: " + directory.toString(false);
    throw std::runtime_error{error};
  }
	std::thread t = std::thread([=]{
		std::vector<Uri> uris(1);
		uris[0] = Uri(path);

		auto provider = ral::io::uri_data_provider(uris);
		auto parser = ral::io::csv_parser(delimiter, line_terminator, skip_rows, names, dtypes);
	  ral::io::data_loader loader(&parser, &provider);

    try
    {
      CodeTimer blazing_timer;
      std::vector<gdf_column_cpp> columns;
      loader.load_data(columns, {}, false);

      blazing_frame output_frame;
      output_frame.add_table(columns);

      double duration = blazing_timer.getDuration();
      
      result_set_repository::get_instance().update_token(token, output_frame, duration);
    } catch (const std::exception& e){
      std::cerr << e.what() << '\n';
      result_set_repository::get_instance().update_token(token, blazing_frame{}, 0.0, e.what());
    }
	});
	t.detach();
	return token;
}

static result_pair loadCsvSchema(uint64_t accessToken, Buffer&& buffer) {
  blazingdb::message::io::LoadCsvFileRequestMessage message(buffer.data());

  std::vector<gdf_dtype> types;
  auto schema = message.fileSchema();
  for(auto val : schema->dtypes)
    types.push_back( (gdf_dtype) val );

  uint64_t resultToken = 0L;
  try {
    bool schema_only = true;
    resultToken = loadCsvAndInsertToResultRepository(schema->path, schema->names, types, schema->delimiter, schema->line_terminator, schema->skip_rows, accessToken, schema_only);
  } catch (const std::exception& e) {
     std::cerr << e.what() << std::endl;
     ResponseErrorMessage errorMessage{ std::string{e.what()} };
     return std::make_pair(Status_Error, errorMessage.getBufferData());
  }
  interpreter::NodeConnectionDTO nodeInfo {
      .port = global_port,
      .path = "/tmp/ral.socket",
      .type = NodeConnectionType {NodeConnectionType_TCP}
  };
  interpreter::ExecutePlanResponseMessage responsePayload{resultToken, nodeInfo};
  return std::make_pair(Status_Success, responsePayload.getBufferData());
}

using result_pair = std::pair<Status, std::shared_ptr<flatbuffers::DetachedBuffer>>;
using FunctionType = result_pair (*)(uint64_t, Buffer&& buffer);

static result_pair closeConnectionService(uint64_t accessToken, Buffer&& requestPayloadBuffer) {
  std::cout << "accessToken: " << accessToken << std::endl;

  try {
    result_set_repository::get_instance().remove_all_connection_tokens(accessToken);
    // NOTE: use next 3 lines to check with "/usr/local/cuda/bin/cuda-memcheck  --leak-check full  ./testing-libgdf"   
    // GDFRefCounter::getInstance()->show_summary();
    // hipDeviceReset();
    // exit(0);
  } catch (const std::exception& e) {
     std::cerr << e.what() << std::endl;
     ResponseErrorMessage errorMessage{ std::string{e.what()} };
     return std::make_pair(Status_Error, errorMessage.getBufferData());
  }

  ZeroMessage response{};
  return std::make_pair(Status_Success, response.getBufferData());
}

static result_pair getResultService(uint64_t accessToken, Buffer&& requestPayloadBuffer) {
  std::cout << "accessToken: " << accessToken << std::endl;

  interpreter::GetResultRequestMessage request(requestPayloadBuffer.data());
  std::cout << "resultToken: " << request.getResultToken() << std::endl;

  try {
    // get result from repository using accessToken and resultToken
    result_set_t result = result_set_repository::get_instance().get_result(accessToken, request.getResultToken());

    
    std::string status = "Error";
    std::string errorMsg = result.errorMsg;
    std::vector<std::string> fieldNames;
    std::vector<uint64_t> columnTokens;
    std::vector<::gdf_dto::gdf_column> values;
    int rows = 0;

    if (errorMsg.empty()) {
      status = "OK";
      //TODO ojo el result siempre es una sola tabla por eso indice 0
      rows =  result.result_frame.get_columns()[0][0].size();


      for(std::size_t i = 0; i < result.result_frame.get_columns()[0].size(); ++i) {
        fieldNames.push_back(result.result_frame.get_columns()[0][i].name());
        columnTokens.push_back(result.result_frame.get_columns()[0][i].get_column_token());

        std::cout << "col_name: " << result.result_frame.get_columns()[0][i].name() << std::endl;
        nvstrings_ipc_transfer ipc;
        gdf_dto::gdf_dtype_extra_info dtype_info;
        ::gdf_dto::gdf_column col;

        std::basic_string<int8_t> data;
        std::basic_string<int8_t> valid;

        if(result.result_frame.get_columns()[0][i].dtype() == GDF_STRING){
          NVStrings* strings = static_cast<NVStrings *> (result.result_frame.get_columns()[0][i].get_gdf_column()->data);
          if(result.result_frame.get_columns()[0][i].size() > 0)
            strings->create_ipc_transfer(ipc);
          dtype_info = gdf_dto::gdf_dtype_extra_info {
                .time_unit = (gdf_dto::gdf_time_unit)0,
            };
          // before
          /*  col = ::gdf_dto::gdf_column {
              .data = data,
              .valid = valid,
              .size = static_cast<gdf_size_type>(result.result_frame.get_columns()[0][i].size()),//.get_gdf_column()->data  ` 
              .dtype = (gdf_dto::gdf_dtype)result.result_frame.get_columns()[0][i].dtype(), // GDF_STRING
              .null_count = static_cast<gdf_size_type>(result.result_frame.get_columns()[0][i].null_count()),
              .dtype_info = dtype_info,
              // custrings data
              .custrings_data = libgdf::ConvertIpcByteArray(ipc)
          };
          */
          col.data = data;
          col.valid = valid;
          col.size = result.result_frame.get_columns()[0][i].size();
          col.dtype =  (gdf_dto::gdf_dtype)result.result_frame.get_columns()[0][i].dtype();
          col.dtype_info = dtype_info;
          // custrings data
          col.custrings_data = libgdf::ConvertIpcByteArray(ipc);

        }else{
          dtype_info = gdf_dto::gdf_dtype_extra_info {
                .time_unit = (gdf_dto::gdf_time_unit)0     // TODO: why is this hardcoded?
          };

          data = libgdf::BuildCudaIpcMemHandler(result.result_frame.get_columns()[0][i].get_gdf_column()->data);
          valid = libgdf::BuildCudaIpcMemHandler(result.result_frame.get_columns()[0][i].get_gdf_column()->valid);
        
        // before
        /*  col = ::gdf_dto::gdf_column {
              .data = data,
              .valid = valid,
              .size = result.result_frame.get_columns()[0][i].size(),
              .dtype = (gdf_dto::gdf_dtype)result.result_frame.get_columns()[0][i].dtype(), 
              .null_count = result.result_frame.get_columns()[0][i].null_count(),
              .dtype_info = dtype_info
          };
        */
          col.data = data;
          col.valid = valid;
          col.size = result.result_frame.get_columns()[0][i].size();
          col.dtype =  (gdf_dto::gdf_dtype)result.result_frame.get_columns()[0][i].dtype();
          col.null_count = result.result_frame.get_columns()[0][i].null_count();
          col.dtype_info = dtype_info;

        }

        values.push_back(col);
      }
    }   

    interpreter::BlazingMetadataDTO  metadata = {
      .status = status,
      .message = errorMsg,
      .time = result.duration,
      .rows = rows
    };

  //  // todo: remove hardcode by creating the resulset vector
  //  gdf_column_cpp column = result.get_columns()[0][0];
  //	std::cout<<"getResultService\n";
  //  print_gdf_column(column.get_gdf_column());
  //  std::cout<<"end:getResultService\n";
  //
  //  auto data = libgdf::BuildCudaIpcMemHandler(column.get_gdf_column()->data);
  //  auto valid = libgdf::BuildCudaIpcMemHandler(column.get_gdf_column()->valid);
  //
  //  std::vector<::gdf_dto::gdf_column> values = {
  //    ::gdf_dto::gdf_column {
  //        .data = data,
  //        .valid = valid,
  //        .size = column.size(),
  //        .dtype = (gdf_dto::gdf_dtype)column.dtype(),
  //        .null_count = column.null_count(),
  //        .dtype_info = gdf_dto::gdf_dtype_extra_info {
  //          .time_unit = (gdf_dto::gdf_time_unit)0,
  //        }
  //    }
  //  };

    interpreter::GetResultResponseMessage responsePayload(metadata, fieldNames, columnTokens, values);
    return std::make_pair(Status_Success, responsePayload.getBufferData());

  } catch (const std::exception& e) {
     std::cerr << e.what() << std::endl;
     ResponseErrorMessage errorMessage{ std::string{e.what()} };
     return std::make_pair(Status_Error, errorMessage.getBufferData());
  } catch (...) {
    ResponseErrorMessage errorMessage{ std::string{"Unknown error"} };
    return std::make_pair(Status_Error, errorMessage.getBufferData());
  }
}

static result_pair freeResultService(uint64_t accessToken, Buffer&& requestPayloadBuffer) {
   std::cout << "freeResultService: " << accessToken << std::endl;

  interpreter::GetResultRequestMessage request(requestPayloadBuffer.data());
  std::cout << "resultToken: " << request.getResultToken() << std::endl;
  bool success = false;
  try {
    success = result_set_repository::get_instance().try_free_result(accessToken, request.getResultToken());
  } catch (const std::runtime_error& e) {
    std::cerr << e.what() << std::endl;
     ResponseErrorMessage errorMessage{ std::string{e.what()} };
     return std::make_pair(Status_Error, errorMessage.getBufferData());
  }
  if(success){
	  ZeroMessage response{};
	  return std::make_pair(Status_Success, response.getBufferData());
  }else{
	  ResponseErrorMessage errorMessage{ std::string{"Could not free result set!"} };
	  return std::make_pair(Status_Error, errorMessage.getBufferData());
  }

}


void load_files(ral::io::data_parser * parser, const std::vector<Uri>& uris, std::vector<gdf_column_cpp>& out_columns) {
	auto provider = ral::io::uri_data_provider(uris);
  ral::io::data_loader loader( parser,&provider);
  loader.load_data(out_columns, {}, true);
}

static result_pair executeFileSystemPlanService (uint64_t accessToken, Buffer&& requestPayloadBuffer) {
  blazingdb::message::io::FileSystemDMLRequestMessage requestPayload(requestPayloadBuffer.data());

  // ExecutePlan
  std::cout << "accessToken: " << accessToken << std::endl;
  std::cout << "query: " << requestPayload.statement << std::endl;
  std::cout << "tableGroup: " << requestPayload.tableGroup.name << std::endl;
 	std::cout << "tables: " << requestPayload.tableGroup.tables.size() << std::endl;
  std::cout << "tableSize: " << requestPayload.tableGroup.tables.size() << std::endl;
	std::cout << "FirstColumn File: "
			<< requestPayload.tableGroup.tables[0].files[0]
			<< std::endl;
  
  uint64_t resultToken = 0L;
  try {
    // Read files
    std::vector<std::vector<gdf_column_cpp>> input_tables;
    std::vector<std::string> table_names;
    std::vector<std::vector<std::string>> all_column_names;
    for(size_t i = 0; i < requestPayload.tableGroup.tables.size(); i++) {
      auto table_info = requestPayload.tableGroup.tables[i];
      std::cout << "\n SchemaType: " << table_info.schemaType << std::endl;
      std::vector<gdf_column_cpp> table_cpp;
      if (table_info.schemaType ==  blazingdb::protocol::io::FileSchemaType_PARQUET) {
        std::vector<Uri> uris;
        for (auto file_path : table_info.files) {
          uris.push_back(Uri{file_path});
        }
        ral::io::parquet_parser parser;
        load_files(&parser, uris, table_cpp);
      } else {
        std::vector<Uri> uris;
        std::transform(table_info.files.begin(), table_info.files.end(),
                      std::back_inserter(uris),
                      [](auto const& file){ return Uri{file}; });
        auto csv_params = table_info.csv;
        std::vector<gdf_dtype> types;
        for(auto val : csv_params.dtypes) {
          types.push_back( (gdf_dtype) val );
        }
        ral::io::csv_parser parser(csv_params.delimiter, csv_params.line_terminator, csv_params.skip_rows, csv_params.names, types);
        load_files(&parser, uris, table_cpp);
      }
      input_tables.push_back(table_cpp);
      table_names.push_back(table_info.name);
      all_column_names.push_back(table_info.columnNames);
    }

    // Execute query
    resultToken = evaluate_query(input_tables, table_names, all_column_names, requestPayload.statement, accessToken, {} );
  } catch (const std::exception& e) {
     std::cerr << e.what() << std::endl;
     ResponseErrorMessage errorMessage{ std::string{e.what()} };
     return std::make_pair(Status_Error, errorMessage.getBufferData());
  }

  interpreter::NodeConnectionDTO nodeInfo {
      .port = global_port,
      .path = "/tmp/ral.socket",
      .type = NodeConnectionType {NodeConnectionType_TCP}
  };
  interpreter::ExecutePlanResponseMessage responsePayload{resultToken, nodeInfo};
  return std::make_pair(Status_Success, responsePayload.getBufferData());
}

static result_pair executePlanService(uint64_t accessToken, Buffer&& requestPayloadBuffer)   {
  interpreter::ExecutePlanRequestMessage requestPayload(requestPayloadBuffer.data());

  // ExecutePlan
  std::cout << "accessToken: " << accessToken << std::endl;
  std::cout << "query: " << requestPayload.getLogicalPlan() << std::endl;
  std::cout << "tableGroup: " << requestPayload.getTableGroup().name << std::endl;
 	std::cout << "tables: " << requestPayload.getTableGroup().tables.size() << std::endl;
  std::cout << "tableSize: " << requestPayload.getTableGroup().tables.size() << std::endl;
	std::cout << "FirstColumnSize: "
			<< requestPayload.getTableGroup().tables[0].columns[0].size
			<< std::endl;
  std::cout << "resultToken: " << requestPayload.getTableGroup().tables[0].resultToken << std::endl;
  //Library::Logging::Logger().logInfo("query:\n" + requestPayload.getLogicalPlan());

  std::vector<void *> handles;
	uint64_t resultToken = 0L;
  try {
    std::tuple<std::vector<std::vector<gdf_column_cpp>>, std::vector<std::string>, std::vector<std::vector<std::string>>> request = libgdf::toBlazingDataframe(accessToken, requestPayload.getTableGroup(),handles);

    resultToken = evaluate_query(std::get<0>(request), std::get<1>(request), std::get<2>(request),
                                        requestPayload.getLogicalPlan(), accessToken, handles);
  } catch (const std::exception& e) {
     std::cerr << e.what() << std::endl;
     ResponseErrorMessage errorMessage{ std::string{e.what()} };
     return std::make_pair(Status_Error, errorMessage.getBufferData());
  }
  interpreter::NodeConnectionDTO nodeInfo {
      .port = global_port,
      .path = "/tmp/ral.socket",
      .type = NodeConnectionType {NodeConnectionType_TCP}
  };
  interpreter::ExecutePlanResponseMessage responsePayload{resultToken, nodeInfo};
  return std::make_pair(Status_Success, responsePayload.getBufferData());
}

static result_pair freeMemoryCallback(uint64_t accessToken, Buffer&& requesBuffer)   {
    FreeMemory::freeAll();
    ZeroMessage response{};
    return std::make_pair(Status_Success, response.getBufferData());
}

static  std::map<int8_t, FunctionType> services;


//@todo execuplan with filesystem
auto  interpreterServices(const blazingdb::protocol::Buffer &requestPayloadBuffer) -> blazingdb::protocol::Buffer {
  RequestMessage request{requestPayloadBuffer.data()};
  std::cout << "header: " << (int)request.messageType() << std::endl;

  auto result = services[request.messageType()] ( request.accessToken(),  request.getPayloadBuffer() );
  ResponseMessage responseObject{result.first, result.second};
  return Buffer{responseObject.getBufferData()};
}


int main(int argc, const char *argv[])
{

  /*#ifndef VERBOSE
  std::cout.rdbuf(nullptr); // substitute internal std::cout buffer with
  #endif // VERBOSE*/
  
    std::cout << "RAL Engine starting" << std::endl;


    FreeMemory::Initialize();

    auto output = new Library::Logging::FileOutput("RAL.log", true);
    Library::Logging::ServiceLogging::getInstance().setLogOutput(output);

    // Init AWS S3 ... TODO see if we need to call shutdown and avoid leaks from s3 percy
    BlazingContext::getInstance()->initExternalSystems();
    
  global_ip = "/tmp/ral.socket";
  //global_port = atoi(port.c_str());

  blazingdb::protocol::UnixSocketConnection connection("/tmp/ral.socket");
  blazingdb::protocol::Server server(connection);

  services.insert(std::make_pair(interpreter::MessageType_ExecutePlan, &executePlanService));
  services.insert(std::make_pair(interpreter::MessageType_ExecutePlanFileSystem, &executeFileSystemPlanService));

  services.insert(std::make_pair(interpreter::MessageType_CloseConnection, &closeConnectionService));
  services.insert(std::make_pair(interpreter::MessageType_GetResult, &getResultService));
  services.insert(std::make_pair(interpreter::MessageType_FreeResult, &freeResultService));
  services.insert(std::make_pair(interpreter::MessageType_RegisterFileSystem, &registerFileSystem));
  services.insert(std::make_pair(interpreter::MessageType_DeregisterFileSystem, &deregisterFileSystem));

  services.insert(std::make_pair(interpreter::MessageType_LoadCsvSchema, &loadCsvSchema));
  services.insert(std::make_pair(interpreter::MessageType_LoadParquetSchema, &loadParquetSchema));

  services.insert(std::make_pair(9, &freeMemoryCallback));

  server.handle(&interpreterServices);

	return 0;
}
