#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : testing-libgdf.cu
 Author      : felipe
 Version     :
 Copyright   : Your copyright notice
 Description : Compute sum of reciprocals using STL on CPU and Thrust on GPU
 ============================================================================
 */

#include <hip/hip_runtime.h>
#include "LogicalFilter.h"
#include "CalciteInterpreter.h"
#include "DataFrame.h"
#include "Utils.cuh"
#include <hip/hip_runtime.h>

#include "ipc/calcite_client.h"

#define BIT_FIVE 0x10
#define BIT_SIX 0x20



void runOriginalTest(){
	gdf_size_type num_elements = 8;

		char * data_left;
		char * data_right;
		char * data_out;
		hipError_t cuda_error = hipMalloc((void **) &data_left,sizeof(int8_t) * num_elements);
		cuda_error = hipMalloc((void **) &data_right,sizeof(int8_t) * num_elements);
		cuda_error = hipMalloc((void **) &data_out,sizeof(int8_t) * num_elements);

		thrust::device_ptr<int8_t> left_ptr= thrust::device_pointer_cast((int8_t *) data_left);
		int8_t int8_value = 2;
	//	thrust::fill(thrust::detail::make_normal_iterator(left_ptr), thrust::detail::make_normal_iterator(left_ptr + num_elements), int8_value);
		thrust::copy(thrust::make_counting_iterator<int8_t>(0),thrust::make_counting_iterator<int8_t>(0) + num_elements, thrust::detail::make_normal_iterator(left_ptr));

		thrust::device_ptr<int8_t> right_ptr= thrust::device_pointer_cast((int8_t *) data_right);
		int8_value = 2;
		thrust::fill(thrust::detail::make_normal_iterator(right_ptr), thrust::detail::make_normal_iterator(right_ptr + num_elements), int8_value);

		//for this simple test we will send in only 8 values
		gdf_valid_type * valid = new gdf_valid_type;


		*valid = 255;
		int num = 4;
		*valid = *valid & ~(1 << num);
		num = 6;
		*valid = *valid & ~(1 << num);

		//shold set lef tand bits 4 and 6 to off

		gdf_valid_type * valid_device;
		cuda_error = hipMalloc((void **) &valid_device,64);
		hipMemcpy(valid_device,valid,sizeof(gdf_valid_type),hipMemcpyHostToDevice);
		gdf_valid_type * valid_out = new gdf_valid_type;
		cuda_error = hipMalloc((void **) &valid_out,1);
		gdf_column lhs;
		gdf_error error = gdf_column_view(&lhs,(void *) data_left, valid_device,num_elements,GDF_INT8);
		lhs.null_count = 2;
		gdf_column rhs;
		error = gdf_column_view(&rhs,(void *) data_right, valid_device,num_elements,GDF_INT8);
		rhs.null_count = 2;
		gdf_column output;
		error = gdf_column_view(&output,(void *) data_out, valid_out,num_elements,GDF_INT8);

		std::cout<<"Left"<<std::endl;
		print_column(&lhs);
		std::cout<<"Right"<<std::endl;
		print_column(&rhs);

		error = gdf_binary_operation_v_v_v(&lhs,&rhs,&output,GDF_EQUAL);

		print_column(&output);

		error = gdf_binary_operation_v_v_v(&lhs,&rhs,&output,GDF_GREATER);

		print_column(&output);

		//copy the data on the host and compare
		thrust::device_ptr<int8_t> out_ptr = thrust::device_pointer_cast((int8_t *) output.data);
		
		gdf_data data = {.ui08=3};
		gdf_scalar three = {data, GDF_UINT8};

		error = gdf_binary_operation_v_s_v(&lhs,&three,&output,GDF_EQUAL);
		error = gdf_binary_operation_v_v_v(&lhs,&rhs,&output,GDF_GREATER);

		print_column(&output);

		gpu_apply_stencil(&lhs, &output, &rhs);

		print_column(&rhs);

	//	hipMemcpy(valid,output.valid,1,hipMemcpyDeviceToHost);

		hipFree(data_left);
		hipFree(data_right);
		hipFree(data_out);
		hipFree(valid_device);
		hipFree(valid_out);

		delete valid;
}

void runInterpreterTest(){
	std::string expression = "AND(=(*($0, $0), 1), =($1, 2))";
	expression = "=(=($1, $0), $0)";

	gdf_column left;
	gdf_column right;
	gdf_column third;

	std::vector<gdf_column *> inputs(3);
	inputs[0] = &left;
	inputs[1] = &right;
	inputs[2] = &third;

	size_t num_values = 32;

	char * input1 = new char[num_values];
	char * input2 = new char[num_values];
	char * input3 = new char[num_values];
	for(int i = 0; i < num_values; i++){
		if(i % 2 == 0){
			input1[i] = 1;
		}else{
			input1[i] = i;
		}
		input2[i] = i;
		input3[i] = 1;
	}

	create_gdf_column(inputs[0], GDF_INT8, num_values, (void *) input1, 1);
	create_gdf_column(inputs[1], GDF_INT8, num_values, (void *) input2, 1);
	create_gdf_column(inputs[2], GDF_INT8, num_values, (void *) input3, 1);

	blazing_frame blzframe;
	blzframe.add_table(inputs);

	gdf_column * output = new gdf_column;
	create_gdf_column(output, GDF_INT8, num_values, nullptr, 1);

	gdf_column * temp = new gdf_column;
	create_gdf_column(temp, GDF_INT8, num_values, nullptr, 1);

	std::cout<<"Input 0 ==>"<<std::endl;
	print_column(inputs[0]);
	std::cout<<std::endl<<"Input 1 ==>"<<std::endl;
	print_column(inputs[1]);
	std::cout<<std::endl<<"Input 2 ==>"<<std::endl;
	print_column(inputs[2]);

	expression = ">($1, 5)";
	std::cout<<"evaluating "<<expression<<std::endl;
	std::cout<<std::endl<<"Output before ==>"<<std::endl;
	print_column(output);
	evaluate_expression(
			blzframe,
			expression,
			output,
			temp);

	std::cout<<std::endl<<"Output after ==>"<<std::endl;
	print_column(output);

	expression = "=(=($1, $0), $2)";
	std::cout<<"evaluating "<<expression<<std::endl;
	std::cout<<std::endl<<"Output before ==>"<<std::endl;
	print_column(output);
	evaluate_expression(
			blzframe,
			expression,
			output,
			temp);

	std::cout<<std::endl<<"Output after ==>"<<std::endl;
	print_column(output);

	delete output;
	delete temp;
	//needs to be able to call this with temp == null to indicate we need to know how much space to allocate for temp
}

void runParquetTest(){
	gdf_column left;
	gdf_column right;
	gdf_column third;

	std::vector<gdf_column *> inputs(3);
	inputs[0] = &left;
	inputs[1] = &right;
	inputs[2] = &third;

	size_t num_values = 100;

	char * input1 = new char[num_values];
	char * input2 = new char[num_values];
	char * input3 = new char[num_values];
	for(int i = 0; i < num_values; i++){
		if(i % 2 == 0){
			input1[i] = 1;
		}else{
			input1[i] = i;
		}
		input2[i] = i;
		input3[i] = 1;
	}

	create_gdf_column(inputs[0], GDF_INT8, num_values, (void *) input1, 1);
	create_gdf_column(inputs[1], GDF_INT8, num_values, (void *) input2, 1);
	create_gdf_column(inputs[2], GDF_INT8, num_values, (void *) input3, 1);

	gdf_column ** columns = new gdf_column*[1];
	columns[0] = &left;
	char * test = (char *)"tester";
	char ** tester = new char *[1];
	tester[0] = test;
	size_t * size = new size_t;
	*size = 100;
	//gdf_error err = read_parquet("/home/felipe/parquet-test.parquet",nullptr,tester,columns,size);
}

void runCalciteTest()
{

	//lets make a simple test where we have three talbes that we join then filter then project
	//this mimics our java code
	std::vector<std::vector<gdf_column *> > input_tables(2);

	std::vector<gdf_column * > hr_emps(3);
	std::vector<gdf_column * > hr_joiner_1(2);
	//std::vector<gdf_column * > hr_joiner_2(2);

	int emps_x[3] = { 1, 2, 3};
	int emps_y[3] = { 4, 5,6};
	int emps_z[3] = { 10, 10, 10};

	hr_emps[0] = new gdf_column;
	create_gdf_column(hr_emps[0], GDF_INT32, 3, (void *) emps_x, 4);
	hr_emps[1] = new gdf_column;
	create_gdf_column(hr_emps[1], GDF_INT32, 3, (void *) emps_y, 4);
	hr_emps[2] = new gdf_column;
	create_gdf_column(hr_emps[2], GDF_INT32, 3, (void *) emps_z, 4);



	int joiner_join_x[6] = { 1, 1, 1, 2, 2, 3};
	int joiner_y[6] = { 1, 2, 3, 4 ,5 ,6};


	hr_joiner_1[0] = new gdf_column;
	create_gdf_column(hr_joiner_1[0], GDF_INT32, 6, (void *) joiner_join_x, 4);

	hr_joiner_1[1] = new gdf_column;
	create_gdf_column(hr_joiner_1[1], GDF_INT32, 6, (void *) joiner_y, 4);

	input_tables[0] = hr_emps;
	input_tables[1] = hr_joiner_1;
	std::vector<std::string> table_names = { "hr.emps" , "hr.joiner"};
	std::vector<std::vector<std::string>> column_names = {{"x","y","z"},{"join_x","join_y"}};

	std::vector<gdf_column *> outputs;
	std::vector<std::string> output_column_names;
	void * temp_space = nullptr; //we arent really using this it seems

	std::string query = "\
LogicalProject(x=[$0], y=[$1], z=[$2], join_x=[$3], y0=[$4], EXPR$6=[+($0, $4)])\n\
  LogicalFilter(condition=[OR(<($0, 5), >($3, 3))])\n\
    LogicalJoin(condition=[OR(=($3, $0), =($3, $1))], joinType=[inner])\n\
      EnumerableTableScan(table=[[hr, emps]])\n\
      EnumerableTableScan(table=[[hr, joiner]]) ";

	gdf_error err = evaluate_query(input_tables, table_names, column_names,
		query, outputs, output_column_names, temp_space);


	for(int i = 0; i < outputs.size(); i++){
		print_column(outputs[i]);
	}
}

int main(void)
{
	//runOriginalTest();
	//runInterpreterTest();
	runCalciteTest();
	//runCalciteClientTest("holas");

	return 0;
}
