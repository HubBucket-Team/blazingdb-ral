#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : testing-libgdf.cu
 Author      : felipe
 Version     :
 Copyright   : Your copyright notice
 Description : MVP
 ============================================================================
 */

#include <hip/hip_runtime.h>
#include <memory>
#include <algorithm>
#include <thread>
#include "CalciteInterpreter.h"
#include "ResultSetRepository.h"
#include "DataFrame.h"
#include "Utils.cuh"
#include "Types.h"
#include <hip/hip_runtime.h>

#include "FreeMemory.h"

#include "gdf_wrapper/gdf_wrapper.cuh"

#include <tuple>

#include <blazingdb/protocol/api.h>
#include <blazingdb/protocol/message/messages.h>
#include <blazingdb/protocol/message/orchestrator/messages.h>
#include <blazingdb/protocol/message/interpreter/messages.h>
#include <blazingdb/protocol/message/io/file_system.h>
#include "ral-message.cuh"


using namespace blazingdb::protocol;

#include <blazingdb/io/Util/StringUtil.h>

#include <blazingdb/io/FileSystem/HadoopFileSystem.h>
#include <blazingdb/io/FileSystem/S3FileSystem.h>
#include <blazingdb/io/FileSystem/FileSystemRepository.h>
#include <blazingdb/io/FileSystem/FileSystemCommandParser.h>
#include <blazingdb/io/FileSystem/FileSystemManager.h>
#include <blazingdb/io/Config/BlazingContext.h>
#include <blazingdb/io/Library/Logging/Logger.h>
#include <blazingdb/io/Library/Logging/CoutOutput.h>
#include <blazingdb/io/Library/Logging/FileOutput.h>
#include "blazingdb/io/Library/Logging/ServiceLogging.h"

#include "CalciteExpressionParsing.h"
#include "io/data_parser/CSVParser.h"
#include "io/data_parser/GDFParser.h"
#include "io/data_parser/ParquetParser.h"
#include "io/data_provider/DummyProvider.h"
#include "io/data_provider/UriDataProvider.h"

#include "io/data_parser/DataParser.h"
#include "io/data_provider/DataProvider.h"
#include "io/DataLoader.h"


#include "CodeTimer.h"
#include "config/BlazingConfig.h"

#include "communication/CommunicationData.h"
#include "communication/factory/MessageFactory.h"
#include "communication/network/Client.h"
#include "communication/network/Server.h"
#include <blazingdb/communication/Context.h>

const Path FS_NAMESPACES_FILE("/tmp/file_system.bin");
using result_pair = std::pair<Status, std::shared_ptr<flatbuffers::DetachedBuffer>>;
using FunctionType = result_pair (*)(uint64_t, Buffer&& buffer);

ConnectionAddress connectionAddress;

static result_pair  registerFileSystem(uint64_t accessToken, Buffer&& buffer) {
  std::cout << "registerFileSystem: " << accessToken << std::endl;
  blazingdb::message::io::FileSystemRegisterRequestMessage message(buffer.data());

  FileSystemConnection fileSystemConnection;
  Path root("/");
  const std::string authority =  message.getAuthority();
  if (message.isLocal()) {
    fileSystemConnection = FileSystemConnection(FileSystemType::LOCAL);
  } else if (message.isHdfs()) {
    auto hdfs = message.getHdfs();
    fileSystemConnection = FileSystemConnection(hdfs.host, hdfs.port, hdfs.user, (HadoopFileSystemConnection::DriverType)hdfs.driverType, hdfs.kerberosTicket);
  } else if (message.isS3()) {
    auto s3 = message.getS3();
    fileSystemConnection = FileSystemConnection(s3.bucketName, ( S3FileSystemConnection::EncryptionType )s3.encryptionType, s3.kmsKeyAmazonResourceName, s3.accessKeyId, s3.secretKey, s3.sessionToken);
  }
  root = message.getRoot();
  if (root.isValid() == false) {
    std::cout << "something went wrong when registering filesystem ..." << std::endl;
    ResponseErrorMessage errorMessage{ std::string{ "ERROR: Invalid root provided when registering file system"} };
    return std::make_pair(Status_Error, errorMessage.getBufferData());
  }
  FileSystemEntity fileSystemEntity(authority, fileSystemConnection, root);
  bool ok = BlazingContext::getInstance()->getFileSystemManager()->deregisterFileSystem(authority);
  ok = BlazingContext::getInstance()->getFileSystemManager()->registerFileSystem(fileSystemEntity);
	if (ok) { // then save the fs
		const FileSystemRepository fileSystemRepository(FS_NAMESPACES_FILE, true);
		const bool saved = fileSystemRepository.add(fileSystemEntity);
		if (saved == false) {
			std::cout << "WARNING: could not save the registered file system into ... the data file uri ..."; //TODO percy error message
		}
	} else {
   	  std::cout << "something went wrong when registering filesystem ..." << std::endl;
      ResponseErrorMessage errorMessage{ std::string{"ERROR: Something went wrong when registering file system"} };
      return std::make_pair(Status_Error, errorMessage.getBufferData());
	}
  ZeroMessage response{};
  return std::make_pair(Status_Success, response.getBufferData());
}

static result_pair  deregisterFileSystem(uint64_t accessToken, Buffer&& buffer)  {
  std::cout << "deregisterFileSystem: " << accessToken << std::endl;
  blazingdb::message::io::FileSystemDeregisterRequestMessage message(buffer.data());
  auto authority =  message.getAuthority();
  if (authority.empty() == true) {
     ResponseErrorMessage errorMessage{ std::string{"derigistering an empty authority"} };
     return std::make_pair(Status_Error, errorMessage.getBufferData());
  }
  const bool ok = BlazingContext::getInstance()->getFileSystemManager()->deregisterFileSystem(authority);
  if (ok) { // then save the fs
    const FileSystemRepository fileSystemRepository(FS_NAMESPACES_FILE, true);
    const bool deleted = fileSystemRepository.deleteByAuthority(authority);
    if (deleted == false) {
      std::cout << "WARNING: could not delete the registered file system into ... the data file uri ..."; //TODO percy error message
    }
  }
  ZeroMessage response{};
  return std::make_pair(Status_Success, response.getBufferData());
}


query_token_t loadParquetAndInsertToResultRepository(std::string path, connection_id_t connection, bool schema_only) {
	std::cout<<"loadParquet\n";

	query_token_t token = result_set_repository::get_instance().register_query(connection); //register the query so we can receive result requests for it
  Path  csvFile(path);
	Uri directory(csvFile.getParentPath().toString());
  const bool existsParquetDir = BlazingContext::getInstance()->getFileSystemManager()->exists(directory);
	const bool isS3Dir = (directory.getFileSystemType() == FileSystemType::S3);
	if ((existsParquetDir == false) && (isS3Dir == false)) {
    auto error = "parquet table folder doesn't exists: " + directory.toString(false);
    throw std::runtime_error{error};
  }

	std::thread t = std::thread([=]{
		std::vector<Uri> uris(1);
		uris[0] = Uri(path);
		
        auto provider = ral::io::uri_data_provider(uris);
        auto parser = ral::io::parquet_parser();
      ral::io::data_loader loader(&parser, &provider);
      
	  try
	  {
	  
	    CodeTimer blazing_timer;
	    std::vector<gdf_column_cpp> columns;
	    loader.load_data(columns, {}, false);

      blazing_frame output_frame;
      output_frame.add_table(columns);

      double duration = blazing_timer.getDuration();

      result_set_repository::get_instance().update_token(token, output_frame, duration);
    } catch(const std::exception& e) {
      std::cerr << e.what() << '\n';
      result_set_repository::get_instance().update_token(token, blazing_frame{}, 0.0, e.what());
    }
	});
	t.detach();
	return token;
}

static result_pair loadParquetSchema(uint64_t accessToken, Buffer&& buffer) {

  blazingdb::message::io::LoadParquetFileRequestMessage message(buffer.data());

  uint64_t resultToken = 0L;
  try {
    // @todo, what about other parameters
    bool schema_only = true;
    resultToken = loadParquetAndInsertToResultRepository(message.fileSchema()->path, accessToken, schema_only);

  } catch (const std::exception& e) {
     std::cerr << e.what() << std::endl;
     ResponseErrorMessage errorMessage{ std::string{e.what()} };
     return std::make_pair(Status_Error, errorMessage.getBufferData());
  }
  
  #ifdef USE_UNIX_SOCKETS

  interpreter::NodeConnectionDTO nodeInfo {
      .port = -1,
      .path = ral::config::BlazingConfig::getInstance().getSocketPath(),
      .type = NodeConnectionType {NodeConnectionType_TCP}
  };

  #else

  interpreter::NodeConnectionDTO nodeInfo {
      .port = connectionAddress.tcp_port,
      .path = ral::config::BlazingConfig::getInstance().getSocketPath(),
      .type = NodeConnectionType {NodeConnectionType_TCP}
  };

  #endif
  
  interpreter::ExecutePlanResponseMessage responsePayload{resultToken, nodeInfo};
  return std::make_pair(Status_Success, responsePayload.getBufferData());
}

query_token_t loadCsvAndInsertToResultRepository(std::string path, std::vector<std::string> names, std::vector<gdf_dtype> dtypes, std::string delimiter, std::string line_terminator, 
  int skip_rows, connection_id_t connection, bool schema_only) {
	std::cout<<"loadCsv\n";

	query_token_t token = result_set_repository::get_instance().register_query(connection); //register the query so we can receive result requests for it
  Path  csvFile(path);
	Uri directory(csvFile.getParentPath().toString());
  const bool existsParquetDir = BlazingContext::getInstance()->getFileSystemManager()->exists(directory);
	const bool isS3Dir = (directory.getFileSystemType() == FileSystemType::S3);
	if ((existsParquetDir == false) && (isS3Dir == false)) {
    auto error = "csv table folder doesn't exists: " + directory.toString(false);
    throw std::runtime_error{error};
  }
	std::thread t = std::thread([=]{
		std::vector<Uri> uris(1);
		uris[0] = Uri(path);

		auto provider = ral::io::uri_data_provider(uris);
		auto parser = ral::io::csv_parser(delimiter, line_terminator, skip_rows, names, dtypes);
	  ral::io::data_loader loader(&parser, &provider);

    try
    {
      CodeTimer blazing_timer;

      size_t num_cols = names.size();
      std::vector<bool> include_column(num_cols, true);


      std::vector<gdf_column_cpp> columns;
      loader.load_data(columns, {}, false);

      blazing_frame output_frame;
      output_frame.add_table(columns);

      double duration = blazing_timer.getDuration();

      result_set_repository::get_instance().update_token(token, output_frame, duration);
    } catch (const std::exception& e){
      std::cerr << e.what() << '\n';
      result_set_repository::get_instance().update_token(token, blazing_frame{}, 0.0, e.what());
    }
	});
	t.detach();
	return token;
}

static result_pair loadCsvSchema(uint64_t accessToken, Buffer&& buffer) {
  blazingdb::message::io::LoadCsvFileRequestMessage message(buffer.data());

  std::vector<gdf_dtype> types;
  auto schema = message.fileSchema();
  for(auto val : schema->dtypes)
    types.push_back( (gdf_dtype) val );

  uint64_t resultToken = 0L;
  try {
    bool schema_only = true;
    resultToken = loadCsvAndInsertToResultRepository(schema->path, schema->names, types, schema->delimiter, schema->line_terminator, schema->skip_rows, accessToken, schema_only);
  } catch (const std::exception& e) {
     std::cerr << e.what() << std::endl;
     ResponseErrorMessage errorMessage{ std::string{e.what()} };
     return std::make_pair(Status_Error, errorMessage.getBufferData());
  }
  
  #ifdef USE_UNIX_SOCKETS

  interpreter::NodeConnectionDTO nodeInfo {
      .port = -1,
      .path = ral::config::BlazingConfig::getInstance().getSocketPath(),
      .type = NodeConnectionType {NodeConnectionType_TCP}
  };

  #else

  interpreter::NodeConnectionDTO nodeInfo {
      .port = connectionAddress.tcp_port,
      .path = ral::config::BlazingConfig::getInstance().getSocketPath(),
      .type = NodeConnectionType {NodeConnectionType_TCP}
  };

  #endif

  interpreter::ExecutePlanResponseMessage responsePayload{resultToken, nodeInfo};
  return std::make_pair(Status_Success, responsePayload.getBufferData());
}

using result_pair = std::pair<Status, std::shared_ptr<flatbuffers::DetachedBuffer>>;
using FunctionType = result_pair (*)(uint64_t, Buffer&& buffer);

static result_pair closeConnectionService(uint64_t accessToken, Buffer&& requestPayloadBuffer) {
  std::cout << "accessToken: " << accessToken << std::endl;

  try {
    result_set_repository::get_instance().remove_all_connection_tokens(accessToken);
    // NOTE: use next 3 lines to check with "/usr/local/cuda/bin/cuda-memcheck  --leak-check full  ./testing-libgdf"
    // GDFRefCounter::getInstance()->show_summary();
    // hipDeviceReset();
    // exit(0);
  } catch (const std::exception& e) {
     std::cerr << e.what() << std::endl;
     ResponseErrorMessage errorMessage{ std::string{e.what()} };
     return std::make_pair(Status_Error, errorMessage.getBufferData());
  }

  ZeroMessage response{};
  return std::make_pair(Status_Success, response.getBufferData());
}

static result_pair getResultService(uint64_t accessToken, Buffer&& requestPayloadBuffer) {
  std::cout << "accessToken: " << accessToken << std::endl;

  interpreter::GetResultRequestMessage request(requestPayloadBuffer.data());
  std::cout << "resultToken: " << request.getResultToken() << std::endl;

  try {
    // get result from repository using accessToken and resultToken
    result_set_t result = result_set_repository::get_instance().get_result(accessToken, request.getResultToken());

    std::string status = "Error";
    std::string errorMsg = result.errorMsg;
    std::vector<std::string> fieldNames;
    std::vector<uint64_t> columnTokens;
    std::vector<::gdf_dto::gdf_column> values;
    int rows = 0;

    if (errorMsg.empty()) {
      status = "OK";
      //TODO ojo el result siempre es una sola tabla por eso indice 0
      rows =  result.result_frame.get_columns()[0][0].size();

      for(std::size_t i = 0; i < result.result_frame.get_columns()[0].size(); ++i) {
        fieldNames.push_back(result.result_frame.get_columns()[0][i].name());
        columnTokens.push_back(result.result_frame.get_columns()[0][i].get_column_token());

        std::cout << "col_name: " << result.result_frame.get_columns()[0][i].name() << std::endl;
        nvstrings_ipc_transfer ipc;
        gdf_dto::gdf_dtype_extra_info dtype_info;
        ::gdf_dto::gdf_column col;

        std::basic_string<int8_t> data;
        std::basic_string<int8_t> valid;

        if(result.result_frame.get_columns()[0][i].dtype() == GDF_STRING){
          NVStrings* strings = static_cast<NVStrings *> (result.result_frame.get_columns()[0][i].get_gdf_column()->data);
          if(result.result_frame.get_columns()[0][i].size() > 0)
            strings->create_ipc_transfer(ipc);
          dtype_info = gdf_dto::gdf_dtype_extra_info {
                .time_unit = (gdf_dto::gdf_time_unit)0,
            };

          col.data = data;
          col.valid = valid;
          col.size = result.result_frame.get_columns()[0][i].size();
          col.dtype =  (gdf_dto::gdf_dtype)result.result_frame.get_columns()[0][i].dtype();
          col.dtype_info = dtype_info;
          col.null_count = static_cast<gdf_size_type>(result.result_frame.get_columns()[0][i].null_count()),
          // custrings data
          col.custrings_data = libgdf::ConvertIpcByteArray(ipc);

        }else{
          dtype_info = gdf_dto::gdf_dtype_extra_info {
                .time_unit = (gdf_dto::gdf_time_unit)0     // TODO: why is this hardcoded?
          };

          data = libgdf::BuildCudaIpcMemHandler(result.result_frame.get_columns()[0][i].get_gdf_column()->data);
          valid = libgdf::BuildCudaIpcMemHandler(result.result_frame.get_columns()[0][i].get_gdf_column()->valid);
        
          col.data = data;
          col.valid = valid;
          col.size = result.result_frame.get_columns()[0][i].size();
          col.dtype =  (gdf_dto::gdf_dtype)result.result_frame.get_columns()[0][i].dtype();
          col.null_count = result.result_frame.get_columns()[0][i].null_count();
          col.dtype_info = dtype_info;
        }

        values.push_back(col);
      }
    }

    interpreter::BlazingMetadataDTO  metadata = {
      .status = status,
      .message = errorMsg,
      .time = result.duration,
      .rows = rows
    };

    interpreter::GetResultResponseMessage responsePayload(metadata, fieldNames, columnTokens, values);
    return std::make_pair(Status_Success, responsePayload.getBufferData());

  } catch (const std::exception& e) {
     std::cerr << e.what() << std::endl;
     ResponseErrorMessage errorMessage{ std::string{e.what()} };
     return std::make_pair(Status_Error, errorMessage.getBufferData());
  } catch (...) {
    ResponseErrorMessage errorMessage{ std::string{"Unknown error"} };
    return std::make_pair(Status_Error, errorMessage.getBufferData());
  }
}

static result_pair freeResultService(uint64_t accessToken, Buffer&& requestPayloadBuffer) {
   std::cout << "freeResultService: " << accessToken << std::endl;

  interpreter::GetResultRequestMessage request(requestPayloadBuffer.data());
  std::cout << "resultToken: " << request.getResultToken() << std::endl;
  bool success = false;
  try {
    success = result_set_repository::get_instance().try_free_result(accessToken, request.getResultToken());
  } catch (const std::runtime_error& e) {
    std::cerr << e.what() << std::endl;
     ResponseErrorMessage errorMessage{ std::string{e.what()} };
     return std::make_pair(Status_Error, errorMessage.getBufferData());
  }
  if(success){
	  ZeroMessage response{};
	  return std::make_pair(Status_Success, response.getBufferData());
  }else{
	  ResponseErrorMessage errorMessage{ std::string{"Could not free result set!"} };
	  return std::make_pair(Status_Error, errorMessage.getBufferData());
  }

}


//TODO: we need to have a centralized place where this can be done
//perhaps a utility in protocol, add bool8 after update
gdf_dtype convert_string_dtype(std::string str){
	if(str == "GDF_INT8"){
		return GDF_INT8;
	}else if(str == "GDF_INT16"){
		return GDF_INT16;
	}else if(str == "GDF_INT32"){
		return GDF_INT32;
	}else if(str == "GDF_INT64"){
		return GDF_INT64;
	}else if(str == "GDF_FLOAT32"){
		return GDF_FLOAT32;
	}else if(str == "GDF_FLOAT64"){
		return GDF_FLOAT64;
	}else if(str == "GDF_DATE32"){
		return GDF_DATE32;
	}else if(str == "GDF_DATE64"){
		return GDF_DATE64;
	}else if(str == "GDF_TIMESTAMP"){
		return GDF_TIMESTAMP;
	}else if(str == "GDF_CATEGORY"){
		return GDF_CATEGORY;
	}else if(str == "GDF_STRING"){
		return GDF_STRING;
	}else if(str == "GDF_STRING_CATEGORY"){
		return GDF_STRING_CATEGORY;
	}else{
		return GDF_invalid;
	}
}


static result_pair parseSchemaService(uint64_t accessToken, Buffer&& requestPayloadBuffer) {
	blazingdb::protocol::orchestrator::DDLCreateTableRequestMessage requestPayload(requestPayloadBuffer.data());

	std::shared_ptr<ral::io::data_parser> parser;
	if(requestPayload.schemaType == blazingdb::protocol::FileSchemaType::FileSchemaType_PARQUET){
		parser = std::make_shared<ral::io::parquet_parser>();

	}else if(requestPayload.schemaType == blazingdb::protocol::FileSchemaType::FileSchemaType_CSV){
		std::vector<gdf_dtype> types;
		for(auto val : requestPayload.columnTypes){
			types.push_back(convert_string_dtype(val));
		}
		parser =  std::make_shared<ral::io::csv_parser>(
				requestPayload.csvDelimiter,
  				requestPayload.csvLineTerminator,
  				(int) requestPayload.csvSkipRows,
  				requestPayload.columnNames, types);
	}else{
		//indicate error here
		//this shoudl be done in the orchestrator
	}

	 std::vector<Uri> uris;
	 for (auto file_path : requestPayload.files) {
	     uris.push_back(Uri{file_path});
	 }

	auto provider = std::make_shared<ral::io::uri_data_provider>(uris);
	auto loader = std::make_shared<ral::io::data_loader>( parser,provider);
	ral::io::Schema schema;
	loader->get_schema(schema);


	blazingdb::protocol::TableSchemaSTL transport_schema = schema.getTransport();

	if(requestPayload.schemaType == blazingdb::protocol::FileSchemaType::FileSchemaType_CSV){
		transport_schema.csvDelimiter = requestPayload.csvDelimiter;
		transport_schema.csvSkipRows = requestPayload.csvSkipRows;
		transport_schema.csvLineTerminator = requestPayload.csvLineTerminator;
	}
	transport_schema.files = requestPayload.files;

	blazingdb::protocol::interpreter::CreateTableResponseMessage responsePayload(transport_schema);
	return std::make_pair(Status_Success, responsePayload.getBufferData());
}

static result_pair executeFileSystemPlanService (uint64_t accessToken, Buffer&& requestPayloadBuffer) {
  blazingdb::message::io::FileSystemDMLRequestMessage requestPayload(requestPayloadBuffer.data());

  //make dataloaders
	std::vector<ral::io::data_loader > input_loaders;
	std::vector<ral::io::Schema> schemas;
	std::vector<std::string> table_names;
  for(auto table : requestPayload.tableGroup().tables){
	  ral::io::Schema schema(table.tableSchema);
	std::shared_ptr<ral::io::data_parser> parser;
	  if(table.schemaType == blazingdb::protocol::FileSchemaType::FileSchemaType_PARQUET){
	  		parser = std::make_shared<ral::io::parquet_parser>();

	  	}else if(table.schemaType == blazingdb::protocol::FileSchemaType::FileSchemaType_CSV){
	  		std::vector<gdf_dtype> types;
			for(auto val : table.tableSchema.types){
				types.push_back((gdf_dtype) val);
			}

	  		parser =  std::make_shared<ral::io::csv_parser>(
	  				table.tableSchema.csvDelimiter,
	  				table.tableSchema.csvLineTerminator,
	  				table.tableSchema.csvSkipRows,
	  				table.tableSchema.names, types);
	  	}else{
	  		parser = std::make_shared<ral::io::gdf_parser>(table,accessToken);
	  	}


	  std::shared_ptr<ral::io::data_provider> provider;
	  std::vector<Uri> uris;
	  	 for (auto file_path : table.tableSchema.files) {
	  	     uris.push_back(Uri{file_path});
	  	 }



	  if(table.schemaType == blazingdb::protocol::FileSchemaType::FileSchemaType_CSV ||
			  table.schemaType == blazingdb::protocol::FileSchemaType::FileSchemaType_PARQUET){
		  	 provider = std::make_shared<ral::io::uri_data_provider>(uris);
	  }else{
		  provider = std::make_shared<ral::io::dummy_data_provider>();
	  }
	  ral::io::data_loader loader( parser,provider);
	  	input_loaders.push_back(loader);
	  	schemas.push_back(schema);
	  	table_names.push_back(table.name);

  }


  std::cout << "accessToken: " << accessToken << std::endl;
  std::cout << "query: " << requestPayload.statement() << std::endl;
  std::cout << "tableGroup: " << requestPayload.tableGroup().name << std::endl;
 	std::cout << "num tables: " << requestPayload.tableGroup().tables.size() << std::endl;
  std::cout << "contextToken: " << requestPayload.communicationContext().token << std::endl;
  std::cout << "contextTotalNodes: " << requestPayload.communicationContext().nodes.size() << std::endl;
  
  uint64_t resultToken = 0L;
  try {

    using blazingdb::communication::ContextToken;
    using blazingdb::communication::Context;
    using blazingdb::communication::Node;
    using blazingdb::communication::Buffer;
    auto& rawCommContext = requestPayload.communicationContext();
    std::vector<std::shared_ptr<Node>> contextNodes;
    for(auto& rawNode: rawCommContext.nodes){
      auto& rawBuffer = rawNode.buffer;
      contextNodes.push_back(Node::Make(Buffer(reinterpret_cast<const char*>(rawBuffer.data()), rawBuffer.size())));
    }
    auto ctxToken = ContextToken::Make(rawCommContext.token);
    Context queryContext{ctxToken, contextNodes, contextNodes[rawCommContext.masterIndex], ""};
    ral::communication::network::Server::getInstance().registerContext(*ctxToken);


    // Execute query
    resultToken = evaluate_query(input_loaders, schemas, table_names, requestPayload.statement(), accessToken, queryContext );

  } catch (const std::exception& e) {
     std::cerr << e.what() << std::endl;
     ResponseErrorMessage errorMessage{ std::string{e.what()} };
     return std::make_pair(Status_Error, errorMessage.getBufferData());
  }

  #ifdef USE_UNIX_SOCKETS

  interpreter::NodeConnectionDTO nodeInfo {
      .port = -1,
      .path = ral::config::BlazingConfig::getInstance().getSocketPath(),
      .type = NodeConnectionType {NodeConnectionType_TCP}
  };

  #else

  interpreter::NodeConnectionDTO nodeInfo {
      .port = connectionAddress.tcp_port,
      .path = ral::config::BlazingConfig::getInstance().getSocketPath(),
      .type = NodeConnectionType {NodeConnectionType_TCP}
  };

  #endif

  interpreter::ExecutePlanResponseMessage responsePayload{resultToken, nodeInfo};
  return std::make_pair(Status_Success, responsePayload.getBufferData());
}

static result_pair executePlanService(uint64_t accessToken, Buffer&& requestPayloadBuffer)   {
  interpreter::ExecutePlanRequestMessage requestPayload(requestPayloadBuffer.data());

  // ExecutePlan
  std::cout << "accessToken: " << accessToken << std::endl;
  std::cout << "query: " << requestPayload.getLogicalPlan() << std::endl;
  std::cout << "tableGroup: " << requestPayload.getTableGroup().name << std::endl;
 	std::cout << "tables: " << requestPayload.getTableGroup().tables.size() << std::endl;
  std::cout << "tableSize: " << requestPayload.getTableGroup().tables.size() << std::endl;
	std::cout << "FirstColumnSize: "
			<< requestPayload.getTableGroup().tables[0].columns[0].size
			<< std::endl;
  std::cout << "resultToken: " << requestPayload.getTableGroup().tables[0].resultToken << std::endl;
  //Library::Logging::Logger().logInfo("query:\n" + requestPayload.getLogicalPlan());

  std::vector<void *> handles;
	uint64_t resultToken = 0L;
  try {
    std::tuple<std::vector<std::vector<gdf_column_cpp>>, std::vector<std::string>, std::vector<std::vector<std::string>>> request = libgdf::toBlazingDataframe(accessToken, requestPayload.getTableGroup(),handles);


    using blazingdb::communication::Context;
    using blazingdb::communication::Node;
    Context queryContext{std::vector<std::shared_ptr<Node>>{}, nullptr, ""};


    resultToken = evaluate_query(std::get<0>(request), std::get<1>(request), std::get<2>(request),
                                        requestPayload.getLogicalPlan(), accessToken, handles, queryContext);
  } catch (const std::exception& e) {
     std::cerr << e.what() << std::endl;
     ResponseErrorMessage errorMessage{ std::string{e.what()} };
     return std::make_pair(Status_Error, errorMessage.getBufferData());
  }
  
  #ifdef USE_UNIX_SOCKETS

  interpreter::NodeConnectionDTO nodeInfo {
      .port = -1,
      .path = ral::config::BlazingConfig::getInstance().getSocketPath(),
      .type = NodeConnectionType {NodeConnectionType_TCP}
  };

  #else

  interpreter::NodeConnectionDTO nodeInfo {
      .port = connectionAddress.tcp_port,
      .path = ral::config::BlazingConfig::getInstance().getSocketPath(),
      .type = NodeConnectionType {NodeConnectionType_TCP}
  };

  #endif

  interpreter::ExecutePlanResponseMessage responsePayload{resultToken, nodeInfo};
  return std::make_pair(Status_Success, responsePayload.getBufferData());
}

static result_pair freeMemoryCallback(uint64_t accessToken, Buffer&& requesBuffer)   {
    FreeMemory::freeAll();
    ZeroMessage response{};
    return std::make_pair(Status_Success, response.getBufferData());
}

static  std::map<int8_t, FunctionType> services;


//@todo execuplan with filesystem
auto  interpreterServices(const blazingdb::protocol::Buffer &requestPayloadBuffer) -> blazingdb::protocol::Buffer {
  RequestMessage request{requestPayloadBuffer.data()};
  std::cout << "header: " << (int)request.messageType() << std::endl;

  auto result = services[request.messageType()] ( request.accessToken(),  request.getPayloadBuffer() );
  ResponseMessage responseObject{result.first, result.second};
  return Buffer{responseObject.getBufferData()};
}


int main(int argc, const char *argv[])
{

    std::cout << "Usage: " << argv[0]
            << " <RAL_ID>"
                " <ORCHESTRATOR_HTTP_COMMUNICATION_[IP|HOSTNAME]> <ORCHESTRATOR_HTTP_COMMUNICATION_PORT>"
                " <RAL_HTTP_COMMUNICATION_[IP|HOSTNAME]> <RAL_HTTP_COMMUNICATION_PORT> <RAL_TCP_PROTOCOL_PORT>" << std::endl;

    if (argc != 7) {
        std::cout << "FATAL: Invalid number of arguments" << std::endl;
        return EXIT_FAILURE;
    }

  // #ifndef VERBOSE
  // std::cout.rdbuf(nullptr); // substitute internal std::cout buffer with
  // #endif // VERBOSE

    std::cout << "RAL Engine starting" << std::endl;

    const std::string ralId = std::string(argv[1]);
    const std::string orchestratorHost = std::string(argv[2]);

    const int orchestratorCommunicationPort = ConnectionUtils::parsePort(argv[3]);

    if (orchestratorCommunicationPort == -1) {
        std::cout << "FATAL: Invalid Orchestrator HTTP communication port " + std::string(argv[3]) << std::endl;
        return EXIT_FAILURE;
    }
    
    const std::string ralHost = std::string(argv[4]);

    const int ralCommunicationPort = ConnectionUtils::parsePort(argv[5]);

    if (ralCommunicationPort == -1) {
        std::cout << "FATAL: Invalid RAL HTTP communication port " + std::string(argv[5]) << std::endl;
        return EXIT_FAILURE;
    }

    const int ralProtocolPort = ConnectionUtils::parsePort(argv[6]);

    if (ralProtocolPort == -1) {
        std::cout << "FATAL: Invalid RAL TCP protocol port " + std::string(argv[6]) << std::endl;
        return EXIT_FAILURE;
    }
    
    auto& communicationData = ral::communication::CommunicationData::getInstance();

    communicationData.initialize(
        std::atoi(ralId.c_str()),
        orchestratorHost,
        orchestratorCommunicationPort,
        ralHost,
        ralCommunicationPort,
        ralProtocolPort);

    std::cout << "RAL ID: " << ralId << std::endl;
    std::cout << "Orchestrator HTTP communication host: " << orchestratorHost << std::endl;
    std::cout << "Orchestrator HTTP communication port: " << orchestratorCommunicationPort << std::endl;
    std::cout << "RAL HTTP communication host: " << ralHost << std::endl;
    std::cout << "RAL HTTP communication port: " << ralCommunicationPort << std::endl;

    try {
        auto nodeDataMesssage = ral::communication::messages::Factory::createNodeDataMessage(communicationData.getSelfNode());
        ral::communication::network::Client::sendNodeData(communicationData.getOrchestratorIp(),
                                                          communicationData.getOrchestratorPort(),
                                                          nodeDataMesssage);

        ral::communication::network::Server::start(ralCommunicationPort);
    } catch (std::exception &e) {
        std::cerr << e.what() << "\n";
        return EXIT_FAILURE;
    }

    auto& config = ral::config::BlazingConfig::getInstance();


#ifdef USE_UNIX_SOCKETS

    config.setLogName("RAL." + ralId + ".log")
          .setSocketPath("/tmp/ral." + ralId + ".socket");

    std::cout << "Socket Name: " << config.getSocketPath() << std::endl;

#else

    // NOTE IMPORTANT PERCY aqui es que pyblazing se entera que este es el ip del RAL en el _send de pyblazing
    config.setLogName("RAL." + ralId + ".log")
          .setSocketPath(ralHost);

    std::cout << "Socket Name: " << config.getSocketPath() << std::endl;

#endif

    std::cout << "Log Name: " << config.getLogName() << std::endl;

    FreeMemory::Initialize();

    auto output = new Library::Logging::FileOutput(config.getLogName(), true);
    Library::Logging::ServiceLogging::getInstance().setLogOutput(output);

    // Init AWS S3 ... TODO see if we need to call shutdown and avoid leaks from s3 percy
    BlazingContext::getInstance()->initExternalSystems();

#ifdef USE_UNIX_SOCKETS

  connectionAddress.unix_socket_path = config.getSocketPath();
  blazingdb::protocol::UnixSocketConnection connection(connectionAddress);

#else

  connectionAddress.tcp_host = "127.0.0.1"; // NOTE always use localhost for protocol server
  connectionAddress.tcp_port = ralProtocolPort;

  std::cout << "RAL TCP protocol port: " << connectionAddress.tcp_port << std::endl;
  
#endif

  blazingdb::protocol::Server server(connectionAddress.tcp_port);

  services.insert(std::make_pair(interpreter::MessageType_ExecutePlanFileSystem, &executeFileSystemPlanService));
  services.insert(std::make_pair(interpreter::MessageType_LoadCsvSchema, &parseSchemaService));
  services.insert(std::make_pair(interpreter::MessageType_CloseConnection, &closeConnectionService));
  services.insert(std::make_pair(interpreter::MessageType_GetResult, &getResultService));
  services.insert(std::make_pair(interpreter::MessageType_FreeResult, &freeResultService));
  services.insert(std::make_pair(interpreter::MessageType_RegisterFileSystem, &registerFileSystem));
  services.insert(std::make_pair(interpreter::MessageType_DeregisterFileSystem, &deregisterFileSystem));

  services.insert(std::make_pair(9, &freeMemoryCallback));

  server.handle(&interpreterServices);

	return 0;
}
