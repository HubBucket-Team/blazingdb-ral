/*
 ============================================================================
 Name        : testing-libgdf.cu
 Author      : felipe
 Version     :
 Copyright   : Your copyright notice
 Description : MVP
 ============================================================================
 */

#include <hip/hip_runtime.h>
#include "CalciteInterpreter.h"
#include "ResultSetRepository.h"
#include "DataFrame.h"
#include "Utils.cuh"
#include "Types.h"

#include "ipc/calcite_client.h"
#include "gdf/gdf.h"

#include <blazingdb/protocol/api.h>
#include <blazingdb/protocol/interpreter/messages.h>

using namespace blazingdb::protocol;

using result_pair = std::pair<Status, std::shared_ptr<flatbuffers::DetachedBuffer>>;
using FunctionType = result_pair (*)(uint64_t, const uint8_t* buffer);

static result_pair closeConnectionService(uint64_t accessToken, const uint8_t* requestPayloadBuffer) {
  std::cout << "accessToken: " << accessToken << std::endl;

  try {
	result_set_repository::get_instance().remove_all_connection_tokens(accessToken);
  } catch (std::runtime_error &error) {
     std::cout << error.what() << std::endl;
     ResponseErrorMessage errorMessage{ std::string{error.what()} };
     return std::make_pair(Status_Error, errorMessage.getBufferData());
  }

  ZeroMessage response{};
  return std::make_pair(Status_Success, response.getBufferData());
}

static result_pair getResultService(uint64_t accessToken, const uint8_t* requestPayloadBuffer) {
   std::cout << "accessToken: " << accessToken << std::endl;

  interpreter::GetResultRequestMessage requestPayload(requestPayloadBuffer);
  std::cout << "resultToken: " << requestPayload.getResultToken() << std::endl;

  // remove from repository using accessToken and resultToken

  ZeroMessage response{}; // @todo: GetResultResponseMessage
  return std::make_pair(Status_Success, response.getBufferData());
}


static result_pair executePlanService(uint64_t accessToken, const uint8_t* requestPayloadBuffer)   {
  interpreter::DMLRequestMessage requestPayload(requestPayloadBuffer);

  std::cout << "accessToken: " << accessToken << std::endl;
  std::cout << "query: " << requestPayload.getLogicalPlan() << std::endl;

  query_token_t resultToken;

  try {

	std::vector<std::vector<gdf_column_cpp> > input_tables;
	std::vector<std::string> table_names;
	std::vector<std::vector<std::string>> column_names;
	std::string logicalPlan = requestPayload.getLogicalPlan();

	resultToken = evaluate_query(input_tables, table_names, column_names,
									logicalPlan, accessToken);

  } catch (std::runtime_error &error) {
     std::cout << error.what() << std::endl;
     ResponseErrorMessage errorMessage{ std::string{error.what()} };
     return std::make_pair(Status_Error, errorMessage.getBufferData());
  }

  interpreter::DMLResponseMessage responsePayload{resultToken};
  return std::make_pair(Status_Success, responsePayload.getBufferData());
}

int main(void)
{ 
	blazingdb::protocol::UnixSocketConnection connection({"/tmp/ral.socket", std::allocator<char>()});
  	blazingdb::protocol::Server server(connection);

	std::map<int8_t, FunctionType> services;
	services.insert(std::make_pair(interpreter::MessageType_ExecutePlan, &executePlanService));
	services.insert(std::make_pair(interpreter::MessageType_CloseConnection, &closeConnectionService));
	services.insert(std::make_pair(interpreter::MessageType_GetResult, &getResultService));
  
	auto interpreterServices = [&services](const blazingdb::protocol::Buffer &requestPayloadBuffer) -> blazingdb::protocol::Buffer {
	  RequestMessage request{requestPayloadBuffer.data()};
	  std::cout << "header: " << (int)request.messageType() << std::endl;
  
	  auto result = services[request.messageType()] ( request.accessToken(),  request.getPayloadBuffer() );
	  ResponseMessage responseObject{result.first, result.second};
	  auto bufferedData = responseObject.getBufferData();
	  Buffer buffer{bufferedData->data(),
					bufferedData->size()};
	  return buffer;
	};
	server.handle(interpreterServices);

	return 0;
}
