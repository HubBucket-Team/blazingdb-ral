/*
 ============================================================================
 Name        : testing-libgdf.cu
 Author      : felipe
 Version     :
 Copyright   : Your copyright notice
 Description : MVP
 ============================================================================
 */

#include <hip/hip_runtime.h>
#include "CalciteInterpreter.h"
#include "ResultSetRepository.h"
#include "DataFrame.h"
#include "Utils.cuh"
#include "Types.h"
#include <hip/hip_runtime.h>

#include "gdf/gdf.h"

#include <tuple>
#include <blazingdb/protocol/api.h>
#include <blazingdb/protocol/message/messages.h>
#include <blazingdb/protocol/message/interpreter/messages.h>
#include "ral-message.cuh"

using namespace blazingdb::protocol;

using result_pair = std::pair<Status, std::shared_ptr<flatbuffers::DetachedBuffer>>;
using FunctionType = result_pair (*)(uint64_t, Buffer&& buffer);

static result_pair closeConnectionService(uint64_t accessToken, Buffer&& requestPayloadBuffer) {
  std::cout << "accessToken: " << accessToken << std::endl;

  try {
	result_set_repository::get_instance().remove_all_connection_tokens(accessToken);
  } catch (std::runtime_error &error) {
     std::cout << error.what() << std::endl;
     ResponseErrorMessage errorMessage{ std::string{error.what()} };
     return std::make_pair(Status_Error, errorMessage.getBufferData());
  }

  ZeroMessage response{};
  return std::make_pair(Status_Success, response.getBufferData());
}

static result_pair getResultService(uint64_t accessToken, Buffer&& requestPayloadBuffer) {
  std::cout << "accessToken: " << accessToken << std::endl;

  interpreter::GetResultRequestMessage request(requestPayloadBuffer.data());
  std::cout << "resultToken: " << request.getResultToken() << std::endl;


  // remove from repository using accessToken and resultToken
  blazing_frame result = result_set_repository::get_instance().get_result(accessToken, request.getResultToken());

  //TODO ojo el result siempre es una sola tabla por eso indice 0
  const int rows = result.get_columns()[0][0].size();

  interpreter::BlazingMetadataDTO  metadata = {
    .status = "OK",
    .message = "metadata message",
    .time = 0.1f,
    .rows = rows
  };

  std::vector<std::string> fieldNames;
  std::vector<::gdf_dto::gdf_column> values;

  //TODO WARNING why 0 why multitables?
  for(int i = 0; i < result.get_columns()[0].size(); ++i) {
	  fieldNames.push_back(result.get_columns()[0][i].name());

	  auto data = libgdf::BuildCudaIpcMemHandler(result.get_columns()[0][i].get_gdf_column()->data);
	  auto valid = libgdf::BuildCudaIpcMemHandler(result.get_columns()[0][i].get_gdf_column()->valid);

	  auto col = ::gdf_dto::gdf_column {
	        .data = data,
	        .valid = valid,
	        .size = result.get_columns()[0][i].size(),
	        .dtype = (gdf_dto::gdf_dtype)result.get_columns()[0][i].dtype(),
	        .null_count = result.get_columns()[0][i].null_count(),
	        .dtype_info = gdf_dto::gdf_dtype_extra_info {
	          .time_unit = (gdf_dto::gdf_time_unit)0,
	        }
	    };

	  values.push_back(col);
  }

//  // todo: remove hardcode by creating the resulset vector
//  gdf_column_cpp column = result.get_columns()[0][0];
//	std::cout<<"getResultService\n";
//  print_gdf_column(column.get_gdf_column());
//  std::cout<<"end:getResultService\n";
//
//  auto data = libgdf::BuildCudaIpcMemHandler(column.get_gdf_column()->data);
//  auto valid = libgdf::BuildCudaIpcMemHandler(column.get_gdf_column()->valid);
//
//  std::vector<::gdf_dto::gdf_column> values = {
//    ::gdf_dto::gdf_column {
//        .data = data,
//        .valid = valid,
//        .size = column.size(),
//        .dtype = (gdf_dto::gdf_dtype)column.dtype(),
//        .null_count = column.null_count(),
//        .dtype_info = gdf_dto::gdf_dtype_extra_info {
//          .time_unit = (gdf_dto::gdf_time_unit)0,
//        }
//    }
//  };

  interpreter::GetResultResponseMessage responsePayload(metadata, fieldNames, values);
  std::cout << "**before return data frame\n" << std::flush;
  return std::make_pair(Status_Success, responsePayload.getBufferData());
}

static result_pair freeResultService(uint64_t accessToken, Buffer&& requestPayloadBuffer) {
   std::cout << "freeResultService: " << accessToken << std::endl;

  interpreter::GetResultRequestMessage request(requestPayloadBuffer.data());
  std::cout << "resultToken: " << request.getResultToken() << std::endl;
  if(result_set_repository::get_instance().free_result(request.getResultToken())){
	  ZeroMessage response{};
	  return std::make_pair(Status_Success, response.getBufferData());
  }else{
	  ResponseErrorMessage errorMessage{ std::string{"Could not free result set!"} };
	  return std::make_pair(Status_Error, errorMessage.getBufferData());
  }

}

static result_pair executePlanService(uint64_t accessToken, Buffer&& requestPayloadBuffer)   {
  interpreter::ExecutePlanRequestMessage requestPayload(requestPayloadBuffer.data());

  // ExecutePlan
  std::cout << "accessToken: " << accessToken << std::endl;
  std::cout << "query: " << requestPayload.getLogicalPlan() << std::endl;
  std::cout << "tableGroup: " << requestPayload.getTableGroup().name << std::endl;
 	std::cout << "tables: " << requestPayload.getTableGroup().tables.size() << std::endl;
  std::cout << "tableSize: " << requestPayload.getTableGroup().tables.size() << std::endl;
	std::cout << "FirstColumnSize: "
			<< requestPayload.getTableGroup().tables[0].columns[0].size
			<< std::endl;
	  std::vector<void *> handles;
	std::tuple<std::vector<std::vector<gdf_column_cpp>>, std::vector<std::string>, std::vector<std::vector<std::string>>> request = libgdf::toBlazingDataframe(requestPayload.getTableGroup(),handles);

  uint64_t resultToken = 0L;
  try {
    resultToken = evaluate_query(std::get<0>(request), std::get<1>(request), std::get<2>(request),
                                        requestPayload.getLogicalPlan(), accessToken,handles);
  } catch (std::exception& error) {
     std::cout << error.what() << std::endl;
     ResponseErrorMessage errorMessage{ std::string{error.what()} };
     return std::make_pair(Status_Error, errorMessage.getBufferData());
  }
  interpreter::NodeConnectionDTO nodeInfo {
      .path = "/tmp/ral.socket",
      .type = NodeConnectionType {NodeConnectionType_IPC}
  };
  interpreter::ExecutePlanResponseMessage responsePayload{resultToken, nodeInfo};
  return std::make_pair(Status_Success, responsePayload.getBufferData());
}

int main(void)
{
	std::cout << "RAL Engine starting"<< std::endl;

  blazingdb::protocol::UnixSocketConnection connection({"/tmp/ral.socket", std::allocator<char>()});
  blazingdb::protocol::Server server(connection);

  std::map<int8_t, FunctionType> services;
  services.insert(std::make_pair(interpreter::MessageType_ExecutePlan, &executePlanService));
  services.insert(std::make_pair(interpreter::MessageType_CloseConnection, &closeConnectionService));
  services.insert(std::make_pair(interpreter::MessageType_GetResult, &getResultService));
  services.insert(std::make_pair(interpreter::MessageType_FreeResult, &freeResultService));

  auto interpreterServices = [&services](const blazingdb::protocol::Buffer &requestPayloadBuffer) -> blazingdb::protocol::Buffer {
    RequestMessage request{requestPayloadBuffer.data()};
    std::cout << "header: " << (int)request.messageType() << std::endl;

    auto result = services[request.messageType()] ( request.accessToken(),  request.getPayloadBuffer() );
    ResponseMessage responseObject{result.first, result.second};
    return Buffer{responseObject.getBufferData()};
  };
  server.handle(interpreterServices);

	return 0;
}
