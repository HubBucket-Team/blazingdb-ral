#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : testing-libgdf.cu
 Author      : felipe
 Version     :
 Copyright   : Your copyright notice
 Description : Compute sum of reciprocals using STL on CPU and Thrust on GPU
 ============================================================================
 */

#include <hip/hip_runtime.h>
#include "LogicalFilter.h"
#include "CalciteInterpreter.h"
#include "DataFrame.h"
#include "Utils.cuh"
#include <hip/hip_runtime.h>

#include "ipc/calcite_client.h"
#include "gdf/gdf.h"

#include <blazingdb/protocol/api.h>

#define BIT_FIVE 0x10
#define BIT_SIX 0x20

struct DMLRequest
{
  char query[1024]; //logicalPlan
};

struct DMLResponse
{
  uint64_t query_token; //token
};

/*void runOriginalTest(){
	gdf_size_type num_elements = 8;

		char * data_left;
		char * data_right;
		char * data_out;
		hipError_t cuda_error = hipMalloc((void **) &data_left,sizeof(int8_t) * num_elements);
		cuda_error = hipMalloc((void **) &data_right,sizeof(int8_t) * num_elements);
		cuda_error = hipMalloc((void **) &data_out,sizeof(int8_t) * num_elements);

		thrust::device_ptr<int8_t> left_ptr= thrust::device_pointer_cast((int8_t *) data_left);
		int8_t int8_value = 2;
	//	thrust::fill(thrust::detail::make_normal_iterator(left_ptr), thrust::detail::make_normal_iterator(left_ptr + num_elements), int8_value);
		thrust::copy(thrust::make_counting_iterator<int8_t>(0),thrust::make_counting_iterator<int8_t>(0) + num_elements, thrust::detail::make_normal_iterator(left_ptr));

		thrust::device_ptr<int8_t> right_ptr= thrust::device_pointer_cast((int8_t *) data_right);
		int8_value = 2;
		thrust::fill(thrust::detail::make_normal_iterator(right_ptr), thrust::detail::make_normal_iterator(right_ptr + num_elements), int8_value);

		//for this simple test we will send in only 8 values
		gdf_valid_type * valid = new gdf_valid_type;


		*valid = 255;
		int num = 4;
		*valid = *valid & ~(1 << num);
		num = 6;
		*valid = *valid & ~(1 << num);

		//shold set lef tand bits 4 and 6 to off

		gdf_valid_type * valid_device;
		cuda_error = hipMalloc((void **) &valid_device,64);
		hipMemcpy(valid_device,valid,sizeof(gdf_valid_type),hipMemcpyHostToDevice);
		gdf_valid_type * valid_out = new gdf_valid_type;
		cuda_error = hipMalloc((void **) &valid_out,1);
		gdf_column lhs;
		gdf_error error = gdf_column_view(&lhs,(void *) data_left, valid_device,num_elements,GDF_INT8);
		lhs.null_count = 2;
		gdf_column rhs;
		error = gdf_column_view(&rhs,(void *) data_right, valid_device,num_elements,GDF_INT8);
		rhs.null_count = 2;
		gdf_column output;
		error = gdf_column_view(&output,(void *) data_out, valid_out,num_elements,GDF_INT8);

		std::cout<<"Left"<<std::endl;
		print_column(&lhs);
		std::cout<<"Right"<<std::endl;
		print_column(&rhs);

		error = gdf_binary_operation_v_v_v(&lhs,&rhs,&output,GDF_EQUAL);

		print_column(&output);

		error = gdf_binary_operation_v_v_v(&lhs,&rhs,&output,GDF_GREATER);

		print_column(&output);

		//copy the data on the host and compare
		thrust::device_ptr<int8_t> out_ptr = thrust::device_pointer_cast((int8_t *) output.data);
		
		gdf_data data = {.ui08=3};
		gdf_scalar three = {data, GDF_UINT8};

		error = gdf_binary_operation_v_s_v(&lhs,&three,&output,GDF_EQUAL);
		error = gdf_binary_operation_v_v_v(&lhs,&rhs,&output,GDF_GREATER);

		print_column(&output);

		gpu_apply_stencil(&lhs, &output, &rhs);

		print_column(&rhs);

	//	hipMemcpy(valid,output.valid,1,hipMemcpyDeviceToHost);

		hipFree(data_left);
		hipFree(data_right);
		hipFree(data_out);
		hipFree(valid_device);
		hipFree(valid_out);

		delete valid;
}*/

/*void runInterpreterTest(){
	std::string expression = "AND(=(*($0, $0), 1), =($1, 2))";
	expression = "=(=($1, $0), $0)";

	gdf_column left;
	gdf_column right;
	gdf_column third;

	std::vector<gdf_column *> inputs(3);
	inputs[0] = &left;
	inputs[1] = &right;
	inputs[2] = &third;

	size_t num_values = 32;

	char * input1 = new char[num_values];
	char * input2 = new char[num_values];
	char * input3 = new char[num_values];
	for(int i = 0; i < num_values; i++){
		if(i % 2 == 0){
			input1[i] = 1;
		}else{
			input1[i] = i;
		}
		input2[i] = i;
		input3[i] = 1;
	}

	create_gdf_column(inputs[0], GDF_INT8, num_values, (void *) input1, 1);
	create_gdf_column(inputs[1], GDF_INT8, num_values, (void *) input2, 1);
	create_gdf_column(inputs[2], GDF_INT8, num_values, (void *) input3, 1);

	blazing_frame blzframe;
	blzframe.add_table(inputs);

	gdf_column * output = new gdf_column;
	create_gdf_column(output, GDF_INT8, num_values, nullptr, 1);

	gdf_column * temp = new gdf_column;
	create_gdf_column(temp, GDF_INT8, num_values, nullptr, 1);

	std::cout<<"Input 0 ==>"<<std::endl;
	print_column(inputs[0]);
	std::cout<<std::endl<<"Input 1 ==>"<<std::endl;
	print_column(inputs[1]);
	std::cout<<std::endl<<"Input 2 ==>"<<std::endl;
	print_column(inputs[2]);

	expression = ">($1, 5)";
	std::cout<<"evaluating "<<expression<<std::endl;
	std::cout<<std::endl<<"Output before ==>"<<std::endl;
	print_column(output);
	evaluate_expression(
			blzframe,
			expression,
			output,
			temp);

	std::cout<<std::endl<<"Output after ==>"<<std::endl;
	print_column(output);

	expression = "=(=($1, $0), $2)";
	std::cout<<"evaluating "<<expression<<std::endl;
	std::cout<<std::endl<<"Output before ==>"<<std::endl;
	print_column(output);
	evaluate_expression(
			blzframe,
			expression,
			output,
			temp);

	std::cout<<std::endl<<"Output after ==>"<<std::endl;
	print_column(output);

	delete output;
	delete temp;
	//needs to be able to call this with temp == null to indicate we need to know how much space to allocate for temp
}*/

/*void runParquetTest(){
	gdf_column left;
	gdf_column right;
	gdf_column third;

	std::vector<gdf_column *> inputs(3);
	inputs[0] = &left;
	inputs[1] = &right;
	inputs[2] = &third;

	size_t num_values = 100;

	char * input1 = new char[num_values];
	char * input2 = new char[num_values];
	char * input3 = new char[num_values];
	for(int i = 0; i < num_values; i++){
		if(i % 2 == 0){
			input1[i] = 1;
		}else{
			input1[i] = i;
		}
		input2[i] = i;
		input3[i] = 1;
	}

	create_gdf_column(inputs[0], GDF_INT8, num_values, (void *) input1, 1);
	create_gdf_column(inputs[1], GDF_INT8, num_values, (void *) input2, 1);
	create_gdf_column(inputs[2], GDF_INT8, num_values, (void *) input3, 1);

	gdf_column ** columns = new gdf_column*[1];
	columns[0] = &left;
	char * test = (char *)"tester";
	char ** tester = new char *[1];
	tester[0] = test;
	size_t * size = new size_t;
	*size = 100;
	//gdf_error err = read_parquet("/home/felipe/parquet-test.parquet",nullptr,tester,columns,size);
}*/

/*void runCalciteTest()
{

	//lets make a simple test where we have three talbes that we join then filter then project
	//this mimics our java code
	std::vector<std::vector<gdf_column *> > input_tables(2);

	std::vector<gdf_column * > hr_emps(3);
	std::vector<gdf_column * > hr_joiner_1(2);
	//std::vector<gdf_column * > hr_joiner_2(2);

	int emps_x[3] = { 1, 2, 3};
	int emps_y[3] = { 4, 5,6};
	int emps_z[3] = { 10, 10, 10};

	hr_emps[0] = new gdf_column;
	create_gdf_column(hr_emps[0], GDF_INT32, 3, (void *) emps_x, 4);
	hr_emps[1] = new gdf_column;
	create_gdf_column(hr_emps[1], GDF_INT32, 3, (void *) emps_y, 4);
	hr_emps[2] = new gdf_column;
	create_gdf_column(hr_emps[2], GDF_INT32, 3, (void *) emps_z, 4);



	int joiner_join_x[6] = { 1, 1, 1, 2, 2, 3};
	int joiner_y[6] = { 1, 2, 3, 4 ,5 ,6};


	hr_joiner_1[0] = new gdf_column;
	create_gdf_column(hr_joiner_1[0], GDF_INT32, 6, (void *) joiner_join_x, 4);

	hr_joiner_1[1] = new gdf_column;
	create_gdf_column(hr_joiner_1[1], GDF_INT32, 6, (void *) joiner_y, 4);

	input_tables[0] = hr_emps;
	input_tables[1] = hr_joiner_1;
	std::vector<std::string> table_names = { "hr.emps" , "hr.joiner"};
	std::vector<std::vector<std::string>> column_names = {{"x","y","z"},{"join_x","join_y"}};

	std::vector<gdf_column *> outputs;
	std::vector<std::string> output_column_names;
	void * temp_space = nullptr; //we arent really using this it seems

	std::string query = "\
LogicalProject(x=[$0], y=[$1], z=[$2], join_x=[$3], y0=[$4], EXPR$6=[+($0, $4)])\n\
  LogicalFilter(condition=[OR(<($0, 5), >($3, 3))])\n\
    LogicalJoin(condition=[OR(=($3, $0), =($3, $1))], joinType=[inner])\n\
      EnumerableTableScan(table=[[hr, emps]])\n\
      EnumerableTableScan(table=[[hr, joiner]]) ";

	gdf_error err = evaluate_query(input_tables, table_names, column_names,
		query, outputs, output_column_names, temp_space);


	for(int i = 0; i < outputs.size(); i++){
		print_column(outputs[i]);
	}
}*/

/*void testStencil(){


	gdf_column * a = new gdf_column;
	gdf_column * b = new gdf_column;
	gdf_column * c = new gdf_column;

	//gdf_column a,b,c;

	int8_t a_data[3] = { 1,2,3 };
	int8_t b_data[3] = { 1,1,0 };
	//create_gdf_column(&a, GDF_INT8, 8, a_data, 1);
	//create_gdf_column(&b, GDF_INT8, 8, b_data, 1);
	//create_gdf_column(&c, GDF_INT8, 8, nullptr, 1);

	create_gdf_column(a, GDF_INT8, 3, a_data, 1);
	create_gdf_column(b, GDF_INT8, 3, b_data, 1);
	create_gdf_column(c, GDF_INT8, 3, nullptr, 1);

//	print_column(&a);
//	print_column(&b);
//	print_column(&c);

	print_column(a);
	print_column(b);
	print_column(c);

//	gpu_apply_stencil(
//						&a,
//						&b,
//						&c
//				);

	gpu_apply_stencil(
						a,
						b,
						c
				);

	//	print_column(&a);
	//	print_column(&b);
	//	print_column(&c);

		print_column(a);
		print_column(b);
		print_column(c);
}*/

typedef uint64_t connection_id;

enum action_t{
  OPEN_CONNECTION,
  CLOSE_CONNECTION
};

enum status_t{
  SUCCESS,
  FAILED
};

struct ConnectionRequest
{
  connection_id connection;
  action_t action;
};

struct ConnectionResponse
{
  status_t action;
  char msg[128];
};

static std::uint8_t data[4096];

int main(void)
{
	blazingdb::protocol::UnixSocketConnection connection("/tmp/socket");
	blazingdb::protocol::Server server(connection);

	auto controller = [](const blazingdb::protocol::Buffer &requestBuffer)
		-> blazingdb::protocol::Buffer {
		const ConnectionRequest *cr = reinterpret_cast<const ConnectionRequest *>(requestBuffer.data());

		ConnectionResponse resp;

		try {
			result_set_repository::get_instance().remove_all_connection_tokens(cr->connection);
			resp.action = SUCCESS;
		}
		catch(const std::runtime_error& e) {
			std::cout<<"Error: "<<e.what()<<std::endl;
			resp.action = FAILED;
			strcpy(resp.msg, e.what());
		}

		std::memcpy(data, &resp, sizeof(ConnectionResponse));

		return blazingdb::protocol::Buffer(data, sizeof(ConnectionResponse));
	};

	server.handle(controller);

	//runOriginalTest();
	//runInterpreterTest();
	//runCalciteTest();

	//testStencil();

	return 0;
}
