#include "hip/hip_runtime.h"

#include <cstdlib>
#include <iostream>
#include <string>
#include <vector>
#include "Interpreter/interpreter_cpp.h"
#include "Interpreter/interpreter_ops.cuh"

#include <CalciteExpressionParsing.h>
#include <CalciteInterpreter.h>
#include <DataFrame.h>
#include <blazingdb/io/Util/StringUtil.h>
#include <gtest/gtest.h>
#include <GDFColumn.cuh>
#include <GDFCounter.cuh>
//#include <Utils.cuh>

#include "gdf/library/scalar.h"
#include "gdf/library/table.h"
#include "gdf/library/table_group.h"
#include "gdf/library/types.h"
#include "gdf/library/api.h"
using namespace gdf::library;

struct EvaluateQueryTest : public ::testing::Test {
  struct InputTestItem {
    std::string query;
    std::string logicalPlan;
    gdf::library::TableGroup tableGroup;
    gdf::library::Table resultTable;
  };

  void CHECK_RESULT(gdf::library::Table& computed_solution,
                    gdf::library::Table& reference_solution) {
        
    computed_solution.print(std::cout);
    reference_solution.print(std::cout);

    for (size_t index = 0; index < reference_solution.size(); index++) {
      const auto& reference_column = reference_solution[index];
      const auto& computed_column = computed_solution[index];
      auto a = reference_column.to_string();
      auto b = computed_column.to_string();
      EXPECT_EQ(a, b);
    }
  }
};

// AUTO GENERATED UNIT TESTS
TEST_F(EvaluateQueryTest, TEST_00) {
  auto input = InputTestItem{
      .query =
          "select c_custkey, c_nationkey, c_acctbal from main.customer where "
          "c_custkey < 15",
      .logicalPlan =
          "LogicalProject(c_custkey=[$0], c_nationkey=[$3], c_acctbal=[$5])\n  "
          "LogicalFilter(condition=[<($0, 15)])\n    "
          "EnumerableTableScan(table=[[main, customer]])",
      .tableGroup =
          LiteralTableGroupBuilder{
              {"main.customer",
               {{"c_custkey",
                 Literals<GDF_INT32>{
                     1,   2,   3,   4,   5,   6,   7,   8,   9,   10,  11,  12,
                     13,  14,  15,  16,  17,  18,  19,  20,  21,  22,  23,  24,
                     25,  26,  27,  28,  29,  30,  31,  32,  33,  34,  35,  36,
                     37,  38,  39,  40,  41,  42,  43,  44,  45,  46,  47,  48,
                     49,  50,  51,  52,  53,  54,  55,  56,  57,  58,  59,  60,
                     61,  62,  63,  64,  65,  66,  67,  68,  69,  70,  71,  72,
                     73,  74,  75,  76,  77,  78,  79,  80,  81,  82,  83,  84,
                     85,  86,  87,  88,  89,  90,  91,  92,  93,  94,  95,  96,
                     97,  98,  99,  100, 101, 102, 103, 104, 105, 106, 107, 108,
                     109, 110, 111, 112, 113, 114, 115, 116, 117, 118, 119, 120,
                     121, 122, 123, 124, 125, 126, 127, 128, 129, 130, 131, 132,
                     133, 134, 135, 136, 137, 138, 139, 140, 141, 142, 143, 144,
                     145, 146, 147, 148, 149, 150}},
                {"c_name",
                 Literals<GDF_INT64>{
                     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0}},
                {"c_address",
                 Literals<GDF_INT64>{
                     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0}},
                {"c_nationkey",
                 Literals<GDF_INT32>{
                     15, 13, 1,  4,  3,  20, 18, 17, 8,  5,  23, 13, 3,  1,
                     23, 10, 2,  6,  18, 22, 8,  3,  3,  13, 12, 22, 3,  8,
                     0,  1,  23, 15, 17, 15, 17, 21, 8,  12, 2,  3,  10, 5,
                     19, 16, 9,  6,  2,  0,  10, 6,  12, 11, 15, 4,  10, 10,
                     21, 13, 1,  12, 17, 7,  21, 3,  23, 22, 9,  12, 9,  22,
                     7,  2,  0,  4,  18, 0,  17, 9,  15, 0,  20, 18, 22, 11,
                     5,  0,  23, 16, 14, 16, 8,  2,  7,  9,  15, 8,  17, 12,
                     15, 20, 2,  19, 9,  10, 10, 1,  15, 5,  16, 10, 22, 19,
                     12, 14, 8,  16, 24, 18, 7,  12, 17, 3,  5,  18, 19, 22,
                     21, 4,  7,  9,  11, 4,  17, 11, 19, 7,  16, 5,  9,  4,
                     1,  9,  16, 1,  13, 3,  18, 11, 19, 18}},
                {"c_phone",
                 Literals<GDF_INT64>{
                     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0}},
                {"c_acctbal",
                 Literals<GDF_FLOAT32>{
                     711.56,  121.65,  7498.12, 2866.83, 794.47,  7638.57,
                     9561.95, 6819.74, 8324.07, 2753.54, -272.6,  3396.49,
                     3857.34, 5266.3,  2788.52, 4681.03, 6.34,    5494.43,
                     8914.71, 7603.4,  1428.25, 591.98,  3332.02, 9255.67,
                     7133.7,  5182.05, 5679.84, 1007.18, 7618.27, 9321.01,
                     5236.89, 3471.53, -78.56,  8589.7,  1228.24, 4987.27,
                     -917.75, 6345.11, 6264.31, 1335.3,  270.95,  8727.01,
                     9904.28, 7315.94, 9983.38, 5744.59, 274.58,  3792.5,
                     4573.94, 4266.13, 855.87,  5630.28, 4113.64, 868.9,
                     4572.11, 6530.86, 4151.93, 6478.46, 3458.6,  2741.87,
                     1536.24, 595.61,  9331.13, -646.64, 8795.16, 242.77,
                     8166.59, 6853.37, 1709.28, 4867.52, -611.19, -362.86,
                     4288.5,  2764.43, 6684.1,  5745.33, 1738.87, 7136.97,
                     5121.28, 7383.53, 2023.71, 9468.34, 6463.51, 5174.71,
                     3386.64, 3306.32, 6327.54, 8031.44, 1530.76, 7354.23,
                     4643.14, 1182.91, 2182.52, 5500.11, 5327.38, 6323.92,
                     2164.48, -551.37, 4088.65, 9889.89, 7470.96, 8462.17,
                     2757.45, -588.38, 9091.82, 3288.42, 2514.15, 2259.38,
                     -716.1,  7462.99, 6505.26, 2953.35, 2912.0,  1027.46,
                     7508.92, 8403.99, 3950.83, 3582.37, 3930.35, 363.75,
                     6428.32, 7865.46, 5897.83, 1842.49, -234.12, 1001.39,
                     9280.71, -986.96, 9127.27, 5073.58, 8595.53, 162.57,
                     2314.67, 4608.9,  8732.91, -842.39, 7838.3,  430.59,
                     7897.78, 9963.15, 6706.14, 2209.81, 2186.5,  6417.31,
                     9748.93, 3328.68, 8071.4,  2135.6,  8959.65, 3849.48}},
                {"c_mktsegment",
                 Literals<GDF_INT64>{
                     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0}},
                {"c_comment",
                 Literals<GDF_INT64>{
                     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                     0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0}}}}}
              .Build(),
      .resultTable =
          LiteralTableBuilder{
              "ResultSet",
              {{"GDF_INT64", Literals<GDF_INT64>{1, 2, 3, 4, 5, 6, 7, 8, 9, 10,
                                                 11, 12, 13, 14}},
               {"GDF_INT64", Literals<GDF_INT64>{15, 13, 1, 4, 3, 20, 18, 17, 8,
                                                 5, 23, 13, 3, 1}},
               {"GDF_FLOAT64",
                Literals<GDF_FLOAT64>{711.56, 121.65, 7498.12, 2866.83, 794.47,
                                      7638.57, 9561.95, 6819.74, 8324.07,
                                      2753.54, -272.6, 3396.49, 3857.34,
                                      5266.3}}}}
              .Build()};

  std::vector<std::vector<gdf_column_cpp> > input_tables = input.tableGroup.ToBlazingFrame();
 
  Table t = TableBuilder{
    "emps",
    {
      { "x", [](Index i) -> DType<GDF_FLOAT64> { return i / 10.0; } },
  	  { "y", [](Index i) -> DType<GDF_INT64> { return i * 1000; } },
    }
  }.Build(150);
   std::vector<gdf_column_cpp> output_columns_cpp = t.ToGdfColumnCpps();

  std::vector<gdf_column *> output_columns(2);
  output_columns[0] = output_columns_cpp[0].get_gdf_column();
  output_columns[1] = output_columns_cpp[1].get_gdf_column();

  std::vector<gdf_column *> input_columns(3);
  input_columns[0] = input_tables[0][0].get_gdf_column();
  input_columns[1] = input_tables[0][1].get_gdf_column();
  input_columns[2] = input_tables[0][2].get_gdf_column();

//step 0:   + * + $0 $1 $2 $1 , + sin $1 2.33   
//step 0:                expr1, expr2   

//step 1:  + * $5 $2 $1 , + $1 $2

//step 2:  + $5 $1 , + $1 $2

// Registers are
// 	0			        1     				2			      3     			  4				      5	    		    6				        n + 3 + 2
// input_col_1, input_col_2, input_col_3, output_col_1, output_col2, processing_1, processing_2 .... processing_n

                                                 // expr1,    expr2:
  std::vector<column_index_type>  left_inputs  = { 0, 5, 5,    1,  5};
  std::vector<column_index_type>  right_inputs = { 1, 2, 1,   -1, -2};
  std::vector<column_index_type>  outputs      = { 5, 5, 3,    5,  4 };


  std::vector<column_index_type>  final_output_positions = { 3 , 4 };

  std::vector<gdf_binary_operator> operators = { GDF_ADD, GDF_MUL, GDF_ADD, GDF_INVALID_BINARY, GDF_ADD};
  std::vector<gdf_unary_operator>  unary_operators = { GDF_INVALID_UNARY,GDF_INVALID_UNARY,GDF_INVALID_UNARY,GDF_SIN,GDF_INVALID_UNARY  };

  using FP32 = gdf::library::GdfEnumType<GDF_FLOAT32>;
    
  gdf::library::Scalar<FP32>  junk_obj;
  junk_obj.setValue(0.0).setValid(true);
  gdf::library::Scalar<FP32> vscalar_obj;
  vscalar_obj.setValue(2.33).setValid(true);

  gdf_scalar junk =  *junk_obj.scalar();
  gdf_scalar scalar_val =  *vscalar_obj.scalar();

  std::vector<gdf_scalar> left_scalars = {junk, junk, junk, junk, junk };
  std::vector<gdf_scalar> right_scalars = {junk, junk ,junk , scalar_val, junk };

  std::vector<column_index_type> new_input_indices = {0, 1, 2 };


  auto error = perform_operation (output_columns, input_columns, left_inputs, right_inputs, outputs, final_output_positions,   operators, unary_operators, left_scalars, right_scalars, new_input_indices);
  ASSERT_EQ(error, GDF_SUCCESS);
}
 