/*
 * Copyright 2018 BlazingDB, Inc.
 *     Copyright 2018 Alexander Ocsa alexander@blazingdb.com
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf.h>

#include <arrow/io/file.h>
#include <arrow/util/logging.h>


#include <parquet/column_writer.h>
#include <parquet/file_writer.h>
#include <parquet/properties.h>
#include <parquet/schema.h>
#include <parquet/types.h>

#include <gtest/gtest.h>
#include <parquet/api/writer.h>
#include <parquet/api/reader.h>

#include "cudf/types.h"

#include "cuio/parquet/util/bit_util.cuh"
#include "cuio/parquet/column_reader.h"
#include "cuio/parquet/file_reader.h"

enum ReaderType : std::uint8_t { kGdf, kParquet };

template <ReaderType T>
struct Readers {};

template <>
struct Readers<kParquet> {
    typedef typename parquet::BoolReader          BoolReader;
    typedef typename parquet::Int32Reader         Int32Reader;
    typedef typename parquet::Int64Reader         Int64Reader;
    typedef typename parquet::FloatReader         FloatReader;
    typedef typename parquet::DoubleReader        DoubleReader;
    typedef typename parquet::ParquetFileReader   FileReader;

    static inline gdf_error init_gdf_buffers(void **host_values, gdf_valid_type** host_valid, int16_t** def_levels, uint32_t values_malloc_size, gdf_size_type column_size){
       *host_values = malloc(values_malloc_size);
       auto n_bytes = gdf::util::PaddedLength(arrow::BitUtil::BytesForBits(column_size));
       *host_valid = (gdf_valid_type*)malloc(n_bytes);
       *def_levels = (int16_t*)malloc(column_size * sizeof(int16_t));
        return GDF_SUCCESS;
    }

    static inline gdf_error buffer_to_gdf_column(gdf_column *output, void *host_values, gdf_valid_type* host_valid, uint32_t values_malloc_size, gdf_size_type column_size, gdf_dtype dtype) {
       void *device_values;
       hipError_t cuda_error = hipMalloc((void **)&device_values, values_malloc_size);
       hipMemcpy(device_values, host_values, values_malloc_size, hipMemcpyHostToDevice);

       gdf_valid_type *device_valid;
       auto n_bytes =  gdf::util::PaddedLength(arrow::BitUtil::BytesForBits(column_size));
       hipMalloc((void **)&device_valid, n_bytes);
       hipMemcpy(device_valid, host_valid, n_bytes, hipMemcpyHostToDevice);

       //auto zero_bits = count_zero_bits(host_valid, column_size);

       free(host_values);
       free(host_valid);
       return gdf_column_view(output, device_values, device_valid, column_size, dtype);
    }
};

 
template <>                                                            
struct Readers<kGdf> {
    typedef typename gdf::parquet::BoolReader   BoolReader;            
    typedef typename gdf::parquet::Int32Reader  Int32Reader;           
    typedef typename gdf::parquet::Int64Reader  Int64Reader;           
    typedef typename gdf::parquet::FloatReader  FloatReader;          
    typedef typename gdf::parquet::DoubleReader DoubleReader;          
    typedef typename gdf::parquet::FileReader   FileReader;            
     

    static inline gdf_error init_gdf_buffers(void **device_values, gdf_valid_type** device_valid, int16_t** def_levels, uint32_t values_malloc_size, gdf_size_type column_size){
        hipError_t cuda_error = hipMalloc(device_values, values_malloc_size);
         auto n_bytes =  gdf::util::PaddedLength(arrow::BitUtil::BytesForBits(column_size));
        hipMalloc(device_valid, n_bytes);
        hipMalloc(def_levels, column_size * sizeof(int16_t));
        return GDF_SUCCESS;
    }
     

    static inline gdf_error buffer_to_gdf_column(gdf_column *output, void *device_values, gdf_valid_type* device_valid, uint32_t values_malloc_size, gdf_size_type column_size, gdf_dtype dtype) {
        return gdf_column_view(output, device_values, device_valid, column_size, dtype);
    }
};

template<parquet::Type::type T>
struct parquet_traits {

};

#define PARQUET_TRAITS_FACTORY(ParquetTypeEnum, ParquetType, GdfTypeValue)  \
    template <> struct parquet_traits<ParquetTypeEnum> {                        \
        typedef ParquetType             parquet_type;                           \
        static inline gdf_dtype         gdf_type() { return GdfTypeValue; }       \
    }

PARQUET_TRAITS_FACTORY(parquet::Type::BOOLEAN, bool, GDF_INT8);
PARQUET_TRAITS_FACTORY(parquet::Type::INT32, int32_t, GDF_INT32);
PARQUET_TRAITS_FACTORY(parquet::Type::INT64, int64_t, GDF_INT64);
PARQUET_TRAITS_FACTORY(parquet::Type::FLOAT, float, GDF_FLOAT32);
PARQUET_TRAITS_FACTORY(parquet::Type::DOUBLE, double, GDF_FLOAT64);

#undef PARQUET_TRAITS_FACTORY


class ParquetReaderAPITest : public testing::Test {
protected:
    ParquetReaderAPITest()
            : filename("/tmp/sample.parquet") {}

    std::int32_t
    genInt32(int i) {
        if (i >= 100 && i < 150) {
            return 10000;
        } else if (i >= 200 && i < 300) {
            return 20000;
        } else if (i >= 310 && i < 350) {
            return 30000;
        } else if (i >= 450 && i < 550) {
            return 40000;
        } else if (i >= 800 && i < 950) {
            return 50000;
        } else {
            return i * 100;
        }
    }

    std::int64_t
    genInt64(int i) {
        if (i >= 100 && i < 150) {
            return 10000;
        } else if (i >= 200 && i < 300) {
            return 20000;
        } else if (i >= 310 && i < 350) {
            return 30000;
        } else if (i >= 450 && i < 550) {
            return 40000;
        } else if (i >= 800 && i < 950) {
            return 50000;
        } else {
            return i * 100000;
        }
    }

    static constexpr std::size_t kGroups       = 1;
    static constexpr std::size_t kRowsPerGroup = 524289;

    void
    SetUp() final {
        try {

            std::shared_ptr<::arrow::io::FileOutputStream> stream;
            PARQUET_THROW_NOT_OK(
                    ::arrow::io::FileOutputStream::Open(filename, &stream));

            std::shared_ptr<::parquet::schema::GroupNode> schema =
                    CreateSchema();

            ::parquet::WriterProperties::Builder builder;
            builder.compression(::parquet::Compression::SNAPPY);
            std::shared_ptr<::parquet::WriterProperties> properties =
                    builder.build();

            std::shared_ptr<::parquet::ParquetFileWriter> file_writer =
                    ::parquet::ParquetFileWriter::Open(stream, schema, properties);

            std::int16_t repetition_level = 0;

            for (std::size_t i = 0; i < kGroups; i++) {
                ::parquet::RowGroupWriter *row_group_writer =
                        file_writer->AppendRowGroup(kRowsPerGroup);

                ::parquet::BoolWriter *bool_writer =
                        static_cast<::parquet::BoolWriter *>(
                                row_group_writer->NextColumn());
                for (std::size_t j = 0; j < kRowsPerGroup; j++) {
                    int ind = i * kRowsPerGroup + j;
                    std::int16_t definition_level = ind % 3 > 0 ? 1 : 0;
                    bool         bool_value       = true;
                    bool_writer->WriteBatch(
                            1, &definition_level, &repetition_level, &bool_value);
                }

                ::parquet::Int32Writer *int32_writer =
                        static_cast<::parquet::Int32Writer *>(
                                row_group_writer->NextColumn());
                for (std::size_t j = 0; j < kRowsPerGroup; j++) {
                    int ind = i * kRowsPerGroup + j;
                    std::int16_t definition_level = ind % 3 > 0 ? 1 : 0;
                    std::int32_t int32_value = genInt32(ind);
                    int32_writer->WriteBatch(
                            1, &definition_level, &repetition_level, &int32_value);
                }

                ::parquet::Int64Writer *int64_writer =
                        static_cast<::parquet::Int64Writer *>(
                                row_group_writer->NextColumn());
                for (std::size_t j = 0; j < kRowsPerGroup; j++) {
                    int ind = i * kRowsPerGroup + j;
                    std::int16_t definition_level = ind % 3 > 0 ? 1 : 0;
                    std::int64_t int64_value = genInt64(ind);
                    int64_writer->WriteBatch(
                            1, &definition_level, &repetition_level, &int64_value);
                }

                ::parquet::DoubleWriter *double_writer =
                        static_cast<::parquet::DoubleWriter *>(
                                row_group_writer->NextColumn());
                for (std::size_t j = 0; j < kRowsPerGroup; j++) {
                    int ind = i * kRowsPerGroup + j;
                    std::int16_t definition_level = ind % 3 > 0 ? 1 : 0;
                    double       double_value     = (double)ind;
                    double_writer->WriteBatch(
                            1, &definition_level, &repetition_level, &double_value);
                }
            }

            file_writer->Close();

            DCHECK(stream->Close().ok());
        } catch (const std::exception &e) {
            FAIL() << "Generate file" << e.what();
        }
    }

    std ::shared_ptr<::parquet::schema::GroupNode>
    CreateSchema() {
        return std::static_pointer_cast<::parquet::schema::GroupNode>(
                ::parquet::schema::GroupNode::Make(
                        "schema",
                        ::parquet::Repetition::REQUIRED,
                        ::parquet::schema::NodeVector{
                                ::parquet::schema::PrimitiveNode::Make(
                                        "boolean_field",
                                        ::parquet::Repetition::OPTIONAL,
                                        ::parquet::Type::BOOLEAN,
                                        ::parquet::LogicalType::NONE),
                                ::parquet::schema::PrimitiveNode::Make(
                                        "int32_field",
                                        ::parquet::Repetition::OPTIONAL,
                                        ::parquet::Type::INT32,
                                        ::parquet::LogicalType::NONE),
                                ::parquet::schema::PrimitiveNode::Make(
                                        "int64_field",
                                        ::parquet::Repetition::OPTIONAL,
                                        ::parquet::Type::INT64,
                                        ::parquet::LogicalType::NONE),
                                ::parquet::schema::PrimitiveNode::Make(
                                        "double_field",
                                        ::parquet::Repetition::OPTIONAL,
                                        ::parquet::Type::DOUBLE,
                                        ::parquet::LogicalType::NONE),
                        }));
    }

    void
    TearDown() final {
        //if (std::remove(filename.c_str())) { FAIL() << "Remove file"; }
    }






    template <ReaderType T, class ColumnReaderType, parquet::Type::type C>
    static inline gdf_error
    convert(gdf_column *column, ColumnReaderType *column_reader, int64_t amount_to_read, uint32_t batch_size) {
        typedef typename parquet_traits<C>::parquet_type    parquet_type;
        parquet_type* values_buffer;
        gdf_valid_type* valid_bits;
        int16_t * definition_level;

        auto values_malloc_size = amount_to_read * sizeof(parquet_type);
        gdf_error status = Readers<T>::init_gdf_buffers((void **)&(values_buffer), &valid_bits, &definition_level, values_malloc_size, amount_to_read);

        std::int64_t levels_read;
        std::int64_t values_read = 0;
        std::int64_t nulls_count;

        int64_t rows_read_total = 0;
        while (column_reader->HasNext() && rows_read_total < amount_to_read) {
            int64_t rows_read = column_reader->ReadBatchSpaced(batch_size,
                                                               &definition_level[rows_read_total],
                                                               nullptr,
                                                               &values_buffer[rows_read_total],
                                                               valid_bits,
                                                               0,
                                                               &levels_read,
                                                               &values_read,
                                                               &nulls_count);
            rows_read_total += rows_read;
        }
        std::cout << "columntype: " << typeid(ColumnReaderType).name() << std::endl;
        Readers<T>::buffer_to_gdf_column(column, (void *)values_buffer, valid_bits, values_malloc_size, amount_to_read, parquet_traits<C>::gdf_type());
        return GDF_SUCCESS;
    }

    template <ReaderType T>
    static inline gdf_error containerFrom(gdf_column *column, std::shared_ptr<parquet::ColumnReader> column_reader, int64_t numRecords, uint32_t batch_size) {

        parquet::Type::type parquetDataType = column_reader->type();

#define WHEN(dataType, Prefix)                                  \
        if ((dataType) == parquetDataType)                          \
            return convert<T, typename Readers<T>::Prefix##Reader, dataType>       \
                    (column, static_cast<typename Readers<T>::Prefix##Reader*>(column_reader.get()), numRecords, batch_size)

        WHEN(parquet::Type::BOOLEAN, Bool);
        WHEN(parquet::Type::INT32, Int32);
        WHEN(parquet::Type::INT64, Int64);
        WHEN(parquet::Type::FLOAT, Float);
        WHEN(parquet::Type::DOUBLE, Double);

#undef WHEN

        std::cout << "ERROR: Bad parquet column type\n";
    }

    template <ReaderType T>
    inline static void
    readRowGroup(const std::unique_ptr<typename Readers<T>::FileReader> &parquet_reader, uint32_t batch_size) {

        std::shared_ptr<parquet::FileMetaData> file_metadata = parquet_reader->metadata();
        const parquet::SchemaDescriptor *schema = file_metadata->schema();
        int numRowGroups = file_metadata->num_row_groups();

        std::vector<gdf_column> columns;

        for (int rowGroupIndex = 0; rowGroupIndex < numRowGroups; rowGroupIndex++) {
            auto groupReader = parquet_reader->RowGroup(rowGroupIndex);
            const parquet::RowGroupMetaData *rowGroupMetadata = groupReader->metadata();
            for (int columnIndex = 0; columnIndex < file_metadata->num_columns(); columnIndex++) {
                const parquet::ColumnDescriptor *column = schema->Column(columnIndex);
                std::unique_ptr<parquet::ColumnChunkMetaData> columnMetaData = rowGroupMetadata->ColumnChunk(
                        columnIndex);
                parquet::Type::type type = column->physical_type();

                if (type != parquet::Type::BYTE_ARRAY){
                    const std::shared_ptr<parquet::ColumnReader> columnReader = groupReader->Column(columnIndex);
                    int64_t numRecords = rowGroupMetadata->num_rows();

                    gdf_column output;
                    containerFrom<T>(&output, columnReader, numRecords, batch_size);
                    columns.push_back(output);
                }
            }
        }
    }
    const std::string filename;

    std::size_t columns_length = 0;
};

TEST_F(ParquetReaderAPITest, ParquetCpp) {
    std::unique_ptr<typename Readers<kParquet>::FileReader> reader = Readers<kParquet>::FileReader::OpenFile(filename);
    
    readRowGroup<kParquet>(reader, kRowsPerGroup);
    
}
TEST_F(ParquetReaderAPITest, CuIOParquetCpp) {
    std::unique_ptr<typename Readers<kGdf>::FileReader> reader = Readers<kGdf>::FileReader::OpenFile(filename);
    
    readRowGroup<kGdf>(reader, kRowsPerGroup);
    
}

 